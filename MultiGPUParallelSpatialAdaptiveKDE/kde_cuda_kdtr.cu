#include "hip/hip_runtime.h"
// Copyright 2016 Guiming Zhang (gzhang45@wisc.edu)
// Distributed under GNU General Public License (GPL) license

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <array>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "SamplePoints.h"
#include "AsciiRaster.h"
#include "Utilities.h"

#include "KDtree.h"
#include "CUDA_KDtree.h"

#include "kde_kernel_kdtr.cu"
//#include "CUDA_KDtree.cu" // it seems this is not needed since we already have CUDA_KDtree.h included

using namespace std;

// distance squared between two points
inline  float Distance2(float x0, float y0, float x1, float y1){
	float dx = x1 - x0;
	float dy = y1 - y0;
	return dx*dx + dy*dy;
}

// mean center of points
void MeanCenter(SamplePoints Points, float &mean_x, float &mean_y);

// (squared) standard distance of points
void StandardDistance2(SamplePoints Points, float &d2);

// bandwidth squared
inline float BandWidth2(SamplePoints Points){
	float d2;
	StandardDistance2(Points, d2);
	return sqrtf(2.0f / (3 * Points.numberOfPoints)) * d2;
}

// Gaussian kernel
inline float GaussianKernel(float h2, float d2){
	if(d2 >= CUT_OFF_FACTOR * h2){
		return 0.0f;
	}
	return expf(d2 / (-2.0f * h2)) / (h2*TWO_PI);
}

//Timothy @ 01/21/2020
//EDIT: Changed AllocateDeviceSamplePoints to return void, and instead utilize pointers to an array
//Changed all functions to utilize the array of pointers
void AllocateDeviceSamplePoints(SamplePoints* dPoints, const SamplePoints Points);
void CopyToDeviceSamplePoints(SamplePoints* dPoints, const SamplePoints hPoints);
void CopyFromDeviceSamplePoints(SamplePoints hPoints, const SamplePoints* dPoints);
SamplePoints AllocateSamplePoints(int n); // random points
SamplePoints ReadSamplePoints(const char *csvFile); // points read from a .csv file
// By Guiming @ 2016-09-04
SamplePoints CopySamplePoints(const SamplePoints Points);
void FreeDeviceSamplePoints(SamplePoints* dPoints);
void FreeSamplePoints(SamplePoints* Points);
void WriteSamplePoints(SamplePoints* Points, const char * csvFile);
void WriteSamplePoints(SamplePoints* Points, float* Hs, float* Ws, const char * csvFile);
void ReformPoints(SamplePoints* dPoints); //Timothy @ 08/13/2021
void DividePoints(SamplePoints* dPoints); //Timothy @ 08/13/2021

void AllocateDeviceAsciiRaster(AsciiRaster* dAscii, const AsciiRaster Ascii);
void CopyToDeviceAsciiRaster(AsciiRaster* dAscii, const AsciiRaster hAscii);
void CopyFromDeviceAsciiRaster(AsciiRaster hAscii, const AsciiRaster dAscii);
AsciiRaster AllocateAsciiRaster(int nCols, int nRows, float xLLCorner, float yLLCorner, float cellSize, float noDataValue);
AsciiRaster ReadAsciiRaster(char * asciiFile); // ascii raster read from a .asc file
AsciiRaster CopyAsciiRaster(const AsciiRaster Ascii);
void FreeDeviceAsciiRaster(AsciiRaster* Ascii);
void FreeAsciiRaster(AsciiRaster* Ascii);
void WriteAsciiRaster(AsciiRaster* Ascii, const char * asciiFile);

float* AllocateEdgeCorrectionWeights(SamplePoints Points); 
void FreeEdgeCorrectionWeights(float* weights);
void ReformECWeights(float** dWeights, float* hWeights); //Timothy @ 08/13/2021

void AllocateDeviceEdgeCorrectionWeights(float** dWeights, SamplePoints Points);
void FreeDeviceEdgeCorrectionWeights(float** weights);

///////// Guiming on 2016-03-16 ///////////////
// the array holding bandwidth at each point
float* AllocateBandwidths(int n); // n is number of points
//Allocation on device now done with pointers instead of return
void AllocateDeviceBandwidths(float** dBandwidths, int n); // n is number of points
void CopyToDeviceBandwidths(float** dBandwidth, const float* hBandwidths, const int n);
void CopyFromDeviceBandwidths(float* hBandwidth, const float* dBandwidths, const int n);
void FreeDeviceBandwidths(float* bandwidths);
void FreeBandwidths(float* bandwidths);
void ReformBandwidths(float** dBand, float* hBand); //Timothy @ 08/13/2021 - Reform bandwidth arrays on host and copy back accross devices

// the array holding inclusive/exclusive density at each point
float* AllocateDen(int n); // n is number of points
void AllocateDeviceDen(float** dDen, int n); // n is number of points
void CopyToDeviceDen(float** dDen, const float* hDen, const int n);
void CopyFromDeviceDen(float* hDen, const float* dDen, const int n);
void CopyDeviceDen(float* dDenTo, const float* dDenFrom, const int n);
void FreeDeviceDen(float** den);
void FreeDen(float* den);
void ReformDensities(float** dDen, float* den); //Timothy @ 12/29/21 - Reforms densities from all devices back into one single array

// compute the optimal Maximum Likelihood Estimation fixed bandwidth
// By Guiming @ 2016-02-26
float MLE_FixedBandWidth(AsciiRaster* Ascii, SamplePoints* Points, float **edgeWeights, float h, float* den0 = NULL, float* den1 = NULL, bool useGPU = false, float** dDen0 = NULL, float** dDen1 = NULL);

// compute fixed bandwidth density at sample points
// By Guiming @ 2016-05-21
void ComputeFixedDensityAtPoints(AsciiRaster Ascii, SamplePoints Points, float *edgeWeights, float h, float* den0 = NULL, float* den1 = NULL, float* dDen0 = NULL, float* dDen1 = NULL);

// compute the log likelihood given single bandwidth h
// By Guiming @ 2016-02-26
float LogLikelihood(AsciiRaster* Ascii, SamplePoints* Points, float **edgeWeights, float h, float* den0 = NULL, float* den1 = NULL, bool useGPU = false, float** dDen0 = NULL, float** dDen1 = NULL);

// compute the log likelihood given bandwidths hs
// By Guiming @ 2016-02-26
// float* den0 : density based on all points, including itself
// float* den1 : leave one out density
float LogLikelihood(AsciiRaster* Ascii, SamplePoints* Points, SamplePoints* gpuPoints, float **edgeWeights, float* hs, float* den0 = NULL, float* den1 = NULL, bool useGPU = false, float** dHs = NULL, float** dDen0 = NULL, float** dDen1 = NULL, float h = 1.0f, float alpha = -0.5f, float** dDen0cpy = NULL);

// compute the log likelihood given a center (h0, alpha0) and step (stepH, stepA)
// By Guiming @ 2016-03-06
void hj_likelihood(AsciiRaster* Ascii, SamplePoints* Points, SamplePoints* gpuPoints, float **edgeWeights, float h0, float alpha0, float stepH, float stepA, int lastdmax, float* logLs, float* hs = NULL, float* den0 = NULL, float* den1 = NULL, bool useGPU = false, float** dHs = NULL, float** dDen0 = NULL, float** dDen1 = NULL, float** dDen0cpy = NULL);

// compute the optimal h and alpha (parameters for calculating the optimal adaptive bandwith)
// By Guiming @ 2016-03-06
void hooke_jeeves(AsciiRaster* Ascii, SamplePoints* Points, SamplePoints* gpuPoints, float **edgeWeights, float h0, float alpha0, float stepH, float stepA, float* optParas, float* hs = NULL, float* den0 = NULL, float* den1 = NULL, bool useGPU = false, float** dHs = NULL, float** dDen0 = NULL, float** dDen1 = NULL, float** dDen0cpy = NULL);

float compGML(float* den0, int n);
///////// Guiming on 2016-03-16 ///////////////


// exact edge effects correction (Diggle 1985)
void EdgeCorrectionWeightsExact(SamplePoints Points, float h, AsciiRaster Ascii, float *weights);
void EdgeCorrectionWeightsExact(SamplePoints Points, float *hs, AsciiRaster Ascii, float *weights);

// check whether the result from sequential computation and that from parallel computation agree
void CheckResults(AsciiRaster AsciiSEQ, AsciiRaster AsciiPARA);

// reduction an array on GPU
void ReductionSumGPU(float* dArray, int numberOfElements);

// extract study area boundary from a raster
// By Guiming @ 2016-09-02
void MarkBoundary(AsciiRaster Ascii, bool useGPU = false);

// compute the closest distances from sample points to study area boundary
// By Guiming @ 2016-09-02
void CalcDist2Boundary(SamplePoints Points, AsciiRaster Ascii, bool useGPU = false);

// sort the sample points on their distances to study area boundary
// By Guiming @ 2016-09-04
void SortSamplePoints(SamplePoints Points);

// comparison function for sort
// By Guiming @ 2016-09-04
int compare ( const void *pa, const void *pb );

void BuildCPUKDtree (SamplePoints Points);
void BuildGPUKDtree ();

void EnableP2P(); //Timothy @ 08/13/2020 - Enable P2P Access Across Devices
void nextDev(int numDev, int& curDev); //Timothy @ 08/14/2020 - Determine next Device to be used
void DevProp(); //Timothy @ 08/24/2020 - Check device properties, primarily for troubleshooting purposes

// By Timothy @ 02/26/2020
//This performs the same tasks as ComputeFixedDensityAtPoints function, however it is designed specifically to run in accross multiple
//GPUs asynchronously
void ComputeFixedDensityDevice(hipStream_t* streams, AsciiRaster* Ascii, SamplePoints* Points, float** edgeWeights, float h, float* den0, float* den1, float** dDen0, float** dDen1);

/* Run in 2 modes
 *
 * Mode 0: Do not read points and mask from files.
 *         User specify # of points and cell size of the estimated intensity surface.
 *         Random points with x, y coordinates in the range [0,100] will be generated.
 *         The cell size (must be less than 100) determines how many cells in the intensity surface raster.
 *
 *         ./kde_cuda [mode] [#points] [cellsize] [skipSEQ] [skipPARA]
 *         e.g., ./kde_cuda 0 100 1.0 0 0
 *
 * Mode 1: Read points and mask from files.
 *
 *         ./kde_cuda [mode] [points_file] [mask_file] [skipSEQ] [skipPARA]
 *         e.g., ./kde_cuda 1 ../Points.csv ../Mask.asc 0 0
 *
*/

/* be very careful with these global variables
 * they are declared in this way to avoid passing additional parameters in functions
*/
KDtree tree; // pointer to the kd tree, can be accessed in any function
CUDA_KDTree GPU_tree[2]; //pointer to the GPU kd tree, can be accessed in any function. EDIT: A copy of the tree 
//is now kept on each GPU with each of these pointers corresponding to a GPU.

vector <Point> dataP; // pointer to the vector to hold data points in kd tree, initilized when building kd tree
float* gpuDen[2]; // this is a global array allocated on gpu to store density values. Used in DensityAtPointsKdtr
//int* gpu_ret_indexes;
//float* gpu_ret_dists;
//float* zeroDen;
int MAX_N_NBRS = 0;

//Timothy @ 08/13/2020
int GPU_N = 1; //Holds number of GPUs on machine
int GPU_C = 0; //Keeps track of our current GPU

hipStream_t streams[2]; //Streams to be used for parallelism

SamplePoints sPoints; // sample of point events

int main(int argc, char *argv[]){
	int NPNTS = 100;                // default # of points
	float CELLSIZE = 1.0f;          // default cellsize
	char* pntFn = "data/Points.csv";  // default points file
	char* maskFn = "data/Mask.asc";   // default mask file
	bool fromFiles = true;          // by default, read Points and Mask from files

	int SKIPSEQ = 0;                // by default, do not skip sequential execution
	int SKIPPARA = 0;               // by default, do not skip parallel execution

	//Guiming May 1, 2016
	int Hoption = 0; // 0 for rule of thumb
					 // 1 for h optimal
					 // 2 for h adaptive
	char* denSEQfn = "data/den_SEQ.asc";
	char* denCUDAfn = "data/den_CUDA.asc";

	// parse commandline arguments
	if(argc != 9){
		printf("Incorrect arguments provided. Exiting...\n");
		printf("Run in mode 0:\n ./kde_cuda 0 #points cellsize h_option skip_sequential skip_parallel denfn_seq, denfn_cuda\n");
		printf("Run in mode 1:\n ./kde_cuda 1 points_file mask_file h_option skip_sequential skip_parallel denfn_seq, denfn_cuda\n");
        return 1;
	}
	else{
		int mode = atoi(argv[1]);
		if(mode == 0){
			fromFiles = false;
			NPNTS = atoi(argv[2]);
			CELLSIZE = (float)atof(argv[3]);
			Hoption = atoi(argv[4]);
			SKIPSEQ = atoi(argv[5]);
			SKIPPARA = atoi(argv[6]);
			denSEQfn = argv[7];
			denCUDAfn = argv[8];
		}
		else if(mode == 1){
			pntFn = argv[2];
			maskFn = argv[3];
			Hoption = atoi(argv[4]);
			SKIPSEQ = atoi(argv[5]);
			SKIPPARA = atoi(argv[6]);
			denSEQfn = argv[7];
			denCUDAfn = argv[8];
		}
		else{
			printf("Incorrect arguments provided. Exiting...\n");
			printf("Run in mode 0:\n ./kde_cuda 0 #points cellsize h_option skip_sequential skip_parallel denfn_seq, denfn_cuda\n");
			printf("Run in mode 1:\n ./kde_cuda 1 points_file mask_file h_option skip_sequential skip_parallel denfn_seq, denfn_cuda\n");
	        return 1;
		}

	}

	//Timothy @ 08/13/2020
	//Assign and print number of Compute Capable Devices
	hipGetDeviceCount(&GPU_N);
	printf("Number of Capable Devices: %d\n", GPU_N);
	printf("Current GPU: %d\n", GPU_C);

	/*for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		DevProp();
	}
	hipSetDevice(0);*/

	//Timothy @ 08/24/2020
	//Enable P2P Access across devices
	//EnableP2P();

	hipError_t error;

	//Timothy @ 12/29/2020
	//Create streams for each available device
	
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		error = hipStreamCreate(&streams[i]);
	}
	if (error != hipSuccess)
	{
		printf("Failed to create streams (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	hipSetDevice(0); //Reset device to first GPU

	//SamplePoints sPoints; // sample of point events
	AsciiRaster Mask;    // a mask indicating the extent of study area
	AsciiRaster DenSurf, DenSurf_CUDA; // the estimated intensity surface
	float *edgeWeights;  // edge effect correct weights (for each point in the sample)
	bool correction = true; // enable edge effect correction
	srand(100); // If not read from files, generate random points

	//Read or generate points
	if (fromFiles){
		sPoints = ReadSamplePoints(pntFn);
		Mask = ReadAsciiRaster(maskFn);
	}
	else{
		sPoints = AllocateSamplePoints(NPNTS);
		Mask = AllocateAsciiRaster(int(100/CELLSIZE), int(100/CELLSIZE), 0.0f, 0.0f, CELLSIZE, -9999.0f);
	}

	DenSurf = CopyAsciiRaster(Mask);

	// parameters
	int numPoints = sPoints.numberOfPoints;
	int nCols = Mask.nCols;
	int nRows = Mask.nRows;
	float xLLCorner = Mask.xLLCorner;
	float yLLCorner = Mask.yLLCorner;
	float noDataValue = Mask.noDataValue;
	float cellSize = Mask.cellSize;

	printf("number of points: %d\n", numPoints);
	printf("cell size: %f\n", cellSize);
	printf("number of cells: %d\n", nCols * nRows);

	printf("skip executing SEQUENTIAL program? %d\n", SKIPSEQ);
	printf("skip executing PARALLEL program? %d\n", SKIPPARA);
	printf("number of threads per block: %d\n", BLOCK_SIZE);

	// do the work
	float cell_x; // x coord of cell
	float cell_y; // y coord of cell
	float p_x;    // x coord of point
	float p_y;    // x coord of point
	float p_w;    // weight of point
	float e_w = 1.0;    // edge effect correction weight

	float h = sqrtf(BandWidth2(sPoints));
	printf("rule of thumb bandwidth h0: %.5f\n", h);

	// timing
	//double start, stop;
	float elaps_seq, elaps_exc, elaps_inc;

	if(SKIPSEQ == 0){

		edgeWeights = NULL;
		edgeWeights = AllocateEdgeCorrectionWeights(sPoints);

	///////////////////////// SEQUENTIAL /////////////////////////////////

		///////////////////////// START CPU TIMING /////////////////////////////
		hipEvent_t startCPU;
		error = hipEventCreate(&startCPU);

		if (error != hipSuccess)
		{
		   printf("Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		hipEvent_t stopCPU;
		error = hipEventCreate(&stopCPU);

		if (error != hipSuccess)
		{
		   printf("Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		// Record the start event
		error = hipEventRecord(startCPU, NULL);
		if (error != hipSuccess)
		{
		   printf("Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}
		///////////////////////// END OF START CPU TIMING /////////////////////////////

		// By Guiming @ 2016-09-11
		MarkBoundary(Mask); // either on GPU or CPU
		CalcDist2Boundary(sPoints, Mask);
		//WriteAsciiRaster(&Mask, "output/boundary.asc");
		SortSamplePoints(sPoints);

		// By Guiming @ 2016-11-03
		BuildCPUKDtree(sPoints);

		float* hs = AllocateBandwidths(numPoints);
		for (int i = 0; i < numPoints; i++) 
		{
			hs[i] = h;
		}

	    // compute edge effect correction weights
		EdgeCorrectionWeightsExact(sPoints, h, Mask, edgeWeights);

		if(Hoption == 1){
			float hopt = MLE_FixedBandWidth(&Mask, &sPoints, &edgeWeights, h, NULL, NULL, false);
			printf("cross validated optimal fixed bandwidth hopt: %.5f\n", hopt);

			for(int i = 0; i < numPoints; i++){
				hs[i] = hopt;
			}

			// update edge correction weights
			if(UPDATEWEIGHTS){
				EdgeCorrectionWeightsExact(sPoints, hs, Mask, edgeWeights);
			}
		}

		if(Hoption == 2){
			float* den0 = AllocateDen(numPoints);
			float* den1 = AllocateDen(numPoints);
			float h0 = h;
			float alpha0 = -0.5;
			float stepH = h0/10;
			float stepA = 0.1;
			float* optParas = (float*)malloc(3 * sizeof(float));

			hooke_jeeves(&Mask, &sPoints, NULL, &edgeWeights, h0, alpha0, stepH, stepA, optParas, hs, den0, den1, false);
			h0 = optParas[0];
			alpha0 = optParas[1];
			float logL = optParas[2];

			if(DEBUG) printf("h0: %.5f alpha0: %.5f Lmax: %.5f\n", h0, alpha0, logL);

			free(optParas);
			optParas = NULL;

			ComputeFixedDensityAtPoints(Mask, sPoints, edgeWeights, h0, den0, NULL, false);
			float gml = compGML(den0, numPoints);
			for(int i = 0; i < numPoints; i++){
				hs[i] = h0 * powf(den0[i]/gml, alpha0);
			}
			FreeDen(den0);
			FreeDen(den1);

			// update edge correction weights
			if(UPDATEWEIGHTS){
				EdgeCorrectionWeightsExact(sPoints, hs, Mask, edgeWeights);
			}
		}

		// KDE
		for (int row = 0; row < nRows; row++){
			cell_y = ROW_TO_YCOORD(row, nRows, yLLCorner, cellSize);
			for (int col = 0; col < nCols; col++){
				cell_x = COL_TO_XCOORD(col, xLLCorner, cellSize);
				int idx = row * nCols + col;
				if (DenSurf.elements[idx] != noDataValue){

					float den = 0.0;
					float hp;
					for (int p = 0; p < numPoints; p++){
						p_x = sPoints.xCoordinates[p];
						p_y = sPoints.yCoordinates[p];
						p_w = sPoints.weights[p];
						hp = hs[p];
						if (correction){
							e_w = edgeWeights[p];
						}
						float d2 = Distance2(p_x, p_y, cell_x, cell_y);
						den += GaussianKernel(hp * hp, d2) * p_w *e_w;
					}
					DenSurf.elements[idx] = den; // intensity, not probability
				}
			}
		}

		///////////////////////// STOP CPU TIMING /////////////////////////////
	    // Record the stop event
	    error = hipEventRecord(stopCPU, NULL);

	    if (error != hipSuccess)
	    {
	        printf("Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }

	    // Wait for the stop event to complete
	    error = hipEventSynchronize(stopCPU);
	    if (error != hipSuccess)
	    {
	        printf("Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }

	    elaps_seq = 0.0f;
	    error = hipEventElapsedTime(&elaps_seq, startCPU, stopCPU);

	    if (error != hipSuccess)
	    {
	        printf("Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }
	    ///////////////////////// END OF STOP CPU TIMING /////////////////////////////
		printf("MAX_N_NBRS=%d\n", MAX_N_NBRS);
		printf("Computation on CPU took %.3f ms\n\n", elaps_seq);

		// write results to file
		WriteAsciiRaster(&DenSurf, denSEQfn);
		WriteSamplePoints(&sPoints, hs, edgeWeights, "pntsSEQ.csv");

		// clean up (only those not needed any more)
		FreeEdgeCorrectionWeights(edgeWeights);
		//FreeAsciiRaster(&DenSurf);
		FreeBandwidths(hs);
	}
////////////////////////// END OF SEQUENTIAL //////////////////////////////

//////////////////////////  CUDA  /////////////////////////////////////////
	if(SKIPPARA == 0){

		//EDIT:Timothy @ 01/29/2021 
		//Changed the way device variables are initialized, allocated, and copied to utilize arrays which identify which GPU
		//the program should be using.
		DenSurf_CUDA = CopyAsciiRaster(Mask);
		SamplePoints dPoints[2]; 
		float* dWeights[2];
		AsciiRaster dAscii[2];

		AllocateDeviceSamplePoints(dPoints, sPoints);
		AllocateDeviceEdgeCorrectionWeights(dWeights, sPoints);
		AllocateDeviceAsciiRaster(dAscii, Mask);

		// Guiming @ 2016-03-17
		float* hs = AllocateBandwidths(sPoints.numberOfPoints);
		float* zeroDen = AllocateDen(sPoints.numberOfPoints);
		for (int i = 0; i < numPoints; i++) {
			hs[i] = h;
			zeroDen[i] = 0.0f;
		}
		
		float* dHs[2];
		AllocateDeviceBandwidths(dHs, sPoints.numberOfPoints);

		float* den0 = AllocateDen(sPoints.numberOfPoints);
		float* dDen0[2]; 
		AllocateDeviceDen(dDen0, sPoints.numberOfPoints);
		float* dDen0cpy[2]; 
		AllocateDeviceDen(dDen0cpy, sPoints.numberOfPoints);

		float* den1 = AllocateDen(sPoints.numberOfPoints);
		float* dDen1[2];
		AllocateDeviceDen(dDen1, sPoints.numberOfPoints);

		AllocateDeviceDen(gpuDen, sPoints.numberOfPoints);
		//gpu_ret_indexes =
		//gpu_ret_dists =

		printf("Allocate DONE...\n"); //DEBUGGING

		///////////////////////// START GPU INCLUSIVE TIMING /////////////////////////////
		hipEvent_t startInc;
		error = hipEventCreate(&startInc);

		if (error != hipSuccess)
		{
		   printf("Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		hipEvent_t stopInc;
		error = hipEventCreate(&stopInc);

		if (error != hipSuccess)
		{
		   printf("Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		// Record the start event
		error = hipEventRecord(startInc, NULL);
		if (error != hipSuccess)
		{
		   printf("Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		///////////////////////// END OF START GPU INCLUSIVE TIMING /////////////////////////////
		CopyToDeviceBandwidths(dHs, hs, sPoints.numberOfPoints);
		int pNum = sPoints.numberOfPoints;

		int NBLOCK_W = (pNum + BLOCK_SIZE - 1) / BLOCK_SIZE;
		int GRID_SIZE_W = (int)(sqrtf(NBLOCK_W)) + 1;
		dim3 dimGrid_W(GRID_SIZE_W, GRID_SIZE_W);

		CopyToDeviceSamplePoints(dPoints, sPoints);
		CopyToDeviceAsciiRaster(dAscii, Mask);

		int cells = dAscii[0].nCols * dAscii[0].nRows;
		CopyToDeviceDen(gpuDen, zeroDen, sPoints.numberOfPoints);

		printf("Copied...\n"); //DEBUGGING

		///////////////////////// START GPU EXCLUSIVE TIMING /////////////////////////////
		hipEvent_t startExc;
		error = hipEventCreate(&startExc);

		if (error != hipSuccess)
		{
		   printf("Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		hipEvent_t stopExc;
		error = hipEventCreate(&stopExc);

		if (error != hipSuccess)
		{
		   printf("Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		// Record the start event
		error = hipEventRecord(startExc, NULL);
		if (error != hipSuccess)
		{
		   printf("Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}
		///////////////////////// END OF START GPU EXLUSIVE TIMING /////////////////////////////

		
		///////////////////////// START SORTING TIMING /////////////////////////////
		hipEvent_t startSort;
		error = hipEventCreate(&startSort);

		if (error != hipSuccess)
		{
		   printf("Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		hipEvent_t stopSort;
		error = hipEventCreate(&stopSort);

		if (error != hipSuccess)
		{
		   printf("Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		// Record the start event
		error = hipEventRecord(startSort, NULL);
		if (error != hipSuccess)
		{
		   printf("Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}
		///////////////////////// END OF START SORTING TIMING /////////////////////////////
		///*
		// By Guiming @ 2016-09-11
		MarkBoundary(dAscii[0], true); // either on GPU or CPU
		CalcDist2Boundary(dPoints[0], dAscii[0], true);
		CopyFromDeviceSamplePoints(sPoints, dPoints);
		SortSamplePoints(sPoints);

		//EDIT:Timothy @ 12/10/2020
		//When adding back sorted points, divide points as they are copied accross GPUs
		CopyToDeviceSamplePoints(dPoints, sPoints);

		///////////////////////// STOP SORTING TIMING /////////////////////////////
	    // Record the stop event
	    error = hipEventRecord(stopSort, NULL);

	    if (error != hipSuccess)
	    {
	        printf("Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }

	    // Wait for the stop event to complete
	    error = hipEventSynchronize(stopSort);
	    if (error != hipSuccess)
	    {
	        printf("Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }

	    float elaps_sort = 0.0f;
	    error = hipEventElapsedTime(&elaps_sort, startSort, stopSort);

	    if (error != hipSuccess)
	    {
	        printf("Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }
	    ///////////////////////// END OF STOP SORTING TIMING /////////////////////////////
		printf("#Sorting took %.3f ms\n", elaps_sort);
	
		printf("Sorted...\n"); //DEBUGGING

		///////////////////////// START KDTREE TIMING /////////////////////////////
		hipEvent_t startKd;
		error = hipEventCreate(&startKd);

		if (error != hipSuccess)
		{
		   printf("Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		hipEvent_t stopKd;
		error = hipEventCreate(&stopKd);

		if (error != hipSuccess)
		{
		   printf("Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		// Record the start event
		error = hipEventRecord(startKd, NULL);
		if (error != hipSuccess)
		{
		   printf("Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}
		///////////////////////// END OF START KDTREE TIMING /////////////////////////////
		
		// By Guiming @ 2016-11-03
		if(SKIPSEQ == 1)
		BuildCPUKDtree(sPoints);
		BuildGPUKDtree(); // needs to build the CPUKDtree first

		///////////////////////// STOP KDTREE TIMING /////////////////////////////
	    // Record the stop event
	    error = hipEventRecord(stopKd, NULL);

	    if (error != hipSuccess)
	    {
	        printf("Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }

	    // Wait for the stop event to complete
	    error = hipEventSynchronize(stopKd);
	    if (error != hipSuccess)
	    {
	        printf("Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }

	    float elaps_kd = 0.0f;
	    error = hipEventElapsedTime(&elaps_kd, startKd, stopKd);

	    if (error != hipSuccess)
	    {
	        printf("Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }
	    ///////////////////////// END OF STOP KDTREE TIMING /////////////////////////////
		printf("#Building kd tree took %.3f ms\n", elaps_kd);
		//EDIT: Timothy @ 12/29/2020
		//Run Kernel Asynchronously accross GPUs
		for (int i = 0; i < GPU_N; i++)
		{
			hipSetDevice(i);
			printf("Current Device: %d\n", i);
			//Alg Step: 1
			CalcEdgeCorrectionWeights<<<dimGrid_W, BLOCK_SIZE, 0, streams[i]>>> (h * h, dPoints[i], dAscii[i], dWeights[i]);
			hipStreamSynchronize(streams[i]);
		}
		hipSetDevice(0); //Reset device to first GPU
		printf("mGPU Kernel...\n"); //DEBUGGING
		hipSetDevice(0); //Reset device to first GPU
		// Guiming @ 2016-03-17
		/////////////////////////////////////////////////////////////////////////////////////////
		int numPoints = sPoints.numberOfPoints;
		if(Hoption == 1){
			float hopt = MLE_FixedBandWidth(dAscii, dPoints, dWeights, h, NULL, den1, true, NULL, dDen1);
			printf("cross validated optimal fixed bandwidth hopt: %.5f\n", hopt);

			// kind of combusome
			//Timothy @ 02/19/2020
			//Running following kernels accross all GPUs
			for (int i = 0; i < GPU_N; i++)
			{
				hipSetDevice(i);
				CalcVaryingBandwidths<<<dimGrid_W, BLOCK_SIZE, 0, streams[i]>>>(dPoints[i], hopt, dHs[i]);
				if (UPDATEWEIGHTS)
				{
					CalcEdgeCorrectionWeights<<<dimGrid_W, BLOCK_SIZE, 0, streams[i]>>>(dHs[i], dPoints[i], dAscii[i], dWeights[i]);
				}
				hipStreamSynchronize(streams[i]);
			}
			hipSetDevice(0); //Reset device to first GPU
		}

		if(Hoption == 2){
			float h0 = h;
			float alpha0 = -0.5;
			float stepH = h0/10;
			float stepA = 0.1;
			float* optParas = (float*)malloc(3 * sizeof(float));
			hooke_jeeves(dAscii, dPoints, dPoints, dWeights, h0, alpha0, stepH, stepA, optParas, hs, den0, den1, true, dHs, dDen0, dDen1, dDen0cpy);
			h0 = optParas[0];
			alpha0 = optParas[1];
			float logL = optParas[2];
			if(DEBUG) printf("h0: %.5f alpha0: %.5f Lmax: %.5f\n", h0, alpha0, logL);
			free(optParas);
			optParas = NULL;

			ComputeFixedDensityDevice(streams, dAscii, dPoints, dWeights, h0, NULL, NULL, dDen0, dDen1);
			
			for (int i = 0; i < GPU_N; i++)
			{
				hipSetDevice(i);
				CopyDeviceDen(dDen0cpy[i], dDen0[i], numPoints);
			}
			hipSetDevice(0);

			//reform points and densities
			ReductionSumGPU(dDen0cpy[0], numPoints);
			
	    	// update bandwidth on GPU
			//Timothy @ 02/19/2020
			//Running following kernels accross all GPUs
			for (int i = 0; i < GPU_N; i++)
			{
				hipSetDevice(i);
				CalcVaryingBandwidths<<<dimGrid_W, BLOCK_SIZE, 0, streams[i]>>>(sPoints, dDen0[i], h0, alpha0, dHs[i]);

				// update weights
				//CopyToDeviceBandwidths(dHs, hs, numPoints);
				if (UPDATEWEIGHTS) {

					CalcEdgeCorrectionWeights <<<dimGrid_W, BLOCK_SIZE, 0, streams[i]>>>(dHs[i], dPoints[i], dAscii[i], dWeights[i]);
				}
				hipStreamSynchronize(streams[i]);
			}
			hipSetDevice(0); //Reset device to first GPU
		}

		//Reform data
		ReformPoints(dPoints);

		hipStreamSynchronize(streams[0]);
		printf("Done...\n\n");
		
		/////////////////////////////////////////////////////////////////////////////////

		// invoke kernel to do density estimation
		int NBLOCK_K = (dAscii[0].nCols*dAscii[0].nRows + BLOCK_SIZE - 1) / BLOCK_SIZE;
	    int GRID_SIZE_K = (int)(sqrtf(NBLOCK_K)) + 1;
	    dim3 dimGrid_K(GRID_SIZE_K, GRID_SIZE_K);

		KernelDesityEstimation<<<dimGrid_K, BLOCK_SIZE, 0, streams[0]>>>(dHs[0], dPoints[0], dAscii[0], dWeights[0]);

		///////////////////////// STOP GPU EXCLUSIVE TIMING /////////////////////////////
	    // Record the stop event
	    error = hipEventRecord(stopExc, NULL);

	    if (error != hipSuccess)
	    {
	        printf("Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }

	    // Wait for the stop event to complete
	    error = hipEventSynchronize(stopExc);
	    if (error != hipSuccess)
	    {
	        printf("Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }

	    elaps_exc = 0.0f;
	    error = hipEventElapsedTime(&elaps_exc, startExc, stopExc);

	    if (error != hipSuccess)
	    {
	        printf("Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }
	    ///////////////////////// END OF STOP GPU EXCLUSIVE TIMING /////////////////////////////

		// copy results back to host
		CopyFromDeviceAsciiRaster(DenSurf_CUDA, dAscii[0]);

		///////////////////////// STOP GPU INCLUSIVE TIMING /////////////////////////////
	    // Record the stop event
	    error = hipEventRecord(stopInc, NULL);

	    if (error != hipSuccess)
	    {
	        printf("Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }

	    // Wait for the stop event to complete
	    error = hipEventSynchronize(stopInc);
	    if (error != hipSuccess)
	    {
	        printf("Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }

	    elaps_inc = 0.0f;
	    error = hipEventElapsedTime(&elaps_inc, startInc, stopInc);

	    if (error != hipSuccess)
	    {
	        printf("Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }
	    ///////////////////////// END OF STOP GPU INCLUSIVE TIMING /////////////////////////////
	    printf("Computation on GPU took %.3f ms (EXCLUSIVE)\n", elaps_exc);
	    printf("Computation on GPU took %.3f ms (INCLUSIVE)\n", elaps_inc);

	    if(SKIPSEQ == 0){
			printf("SPEEDUP: %.3fx (EXCLUSIVE) %.3fx (INCLUSIVE)\n", elaps_seq / elaps_exc, elaps_seq / elaps_inc);
			// check resutls
			CheckResults(DenSurf, DenSurf_CUDA);
		}
		// write results to file
		WriteAsciiRaster(&DenSurf_CUDA, denCUDAfn);
		WriteSamplePoints(&sPoints, "pntsCUDA.csv");

		printf("Begin cleanup...\n"); //DEBUGGING
		// clean up
		FreeDeviceSamplePoints(dPoints);
		FreeDeviceEdgeCorrectionWeights(dWeights);
		FreeDeviceAsciiRaster(dAscii);
		FreeSamplePoints(&sPoints);
		// By Guiming @ 2016-09-02
		free(sPoints.distances);
		sPoints.distances = NULL;

		FreeAsciiRaster(&DenSurf);
		FreeAsciiRaster(&DenSurf_CUDA);
		FreeAsciiRaster(&Mask);
		FreeAsciiRaster(dAscii);
		FreeBandwidths(hs);
		FreeDeviceBandwidths(dHs[0]);
		FreeDen(den0);
		FreeDeviceDen(dDen0);
		FreeDeviceDen(dDen0cpy);
		FreeDen(den1);
		FreeDeviceDen(dDen1);
		FreeDen(zeroDen);
		FreeDeviceDen(gpuDen);
	}

	printf("MAX_N_NBRS=%d\n", MAX_N_NBRS);
	printf("Done...\n\n");

	return 0;
}

// mean center of points
void MeanCenter(SamplePoints Points, float &mean_x, float& mean_y){
	float sum_x = 0.0;
	float sum_y = 0.0;

	for (int p = 0; p < Points.numberOfPoints; p++){
		sum_x += Points.xCoordinates[p];
		sum_y += Points.yCoordinates[p];
	}

	mean_x = sum_x / Points.numberOfPoints;
	mean_y = sum_y / Points.numberOfPoints;
}

// standard distance squared
void StandardDistance2(SamplePoints Points, float &d2){

	float mean_x, mean_y;
	MeanCenter(Points, mean_x, mean_y);

	float sum2 = 0.0;

	for (int p = 0; p < Points.numberOfPoints; p++){
		sum2 += Distance2(mean_x, mean_y, Points.xCoordinates[p], Points.yCoordinates[p]);
	}

	d2 = sum2 / Points.numberOfPoints;
}

// generate random sample points
SamplePoints AllocateSamplePoints(int n){
	SamplePoints Points;

	Points.numberOfPoints = n;
	Points.start = 0;
	Points.end = n;
	int size = n*sizeof(float);

	Points.xCoordinates = (float*)malloc(size);
	Points.yCoordinates = (float*)malloc(size);
	Points.weights = (float*)malloc(size);
	Points.distances = (float*)malloc(size); // By Guiming @ 2016-09-02

	for (int i = 0; i < n; i++)
	{
		Points.xCoordinates[i] = rand() * 100.0f / RAND_MAX;
		Points.yCoordinates[i] = rand() * 100.0f / RAND_MAX;
		Points.weights[i] = 1.0f;
		Points.distances[i] = 0.0f; // By Guiming @ 2016-09-02
		//printf("x:%.2f y:%.2f w:%.2f\n", Points.xCoordinates[i], Points.yCoordinates[i], Points.weights[i]);
	}
	return Points;
}

// points read from a .csv file
SamplePoints ReadSamplePoints(const char *csvFile){
	FILE *f = fopen(csvFile, "rt");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	const int CSV_LINE_LENGTH = 256;
	SamplePoints Points;
	int n = 0;
	char line[CSV_LINE_LENGTH];
	char ch;

	while (!feof(f))
	{
		ch = fgetc(f);
		if (ch == '\n')
		{
			n++;
		}
	}

	if (n == 1){
		printf("No point in file!\n");
		exit(1);
	}

	n = n - 1; // do not count the header line
	Points.numberOfPoints = n;
	Points.xCoordinates = (float*)malloc(n*sizeof(float));
	Points.yCoordinates = (float*)malloc(n*sizeof(float));
	Points.weights = (float*)malloc(n*sizeof(float));
	Points.distances = (float*)malloc(n*sizeof(float)); // By Guiming @ 2016-09-02

	int counter = 0;
	char * pch;
	float x, y;
	rewind(f); // go back to the beginning of file
	fgets(line, CSV_LINE_LENGTH, f); //skip the header line
	while (fgets(line, CSV_LINE_LENGTH, f) != NULL){
		pch = strtok(line, ",\n");
		x = atof(pch);
		while (pch != NULL)
		{
			pch = strtok(NULL, ",\n");
			y = atof(pch);
			break;
		}
		Points.xCoordinates[counter] = x;
		Points.yCoordinates[counter] = y;
		Points.weights[counter] = 1.0f;
		Points.distances[counter] = 0.0f; // By Guiming @ 2016-09-02

		counter++;
	}

	fclose(f);

	return Points;
}

void AllocateDeviceSamplePoints(SamplePoints* dPoints, const SamplePoints Points){
	//Timothy @ 01/15/2021
	//EDIT: Changing dPoints to be a array of pointers to each set of points on each device.
	for (int i = 0; i < GPU_N; i++)
	{
		dPoints[i] = Points;
	
		dPoints[i].numberOfPoints = Points.numberOfPoints;
		dPoints[i].totalNumPoints = Points.numberOfPoints;
		int size = Points.numberOfPoints * sizeof(float);
		hipError_t error;

		hipSetDevice(i);
		error = hipMalloc((void**)&dPoints[i].xCoordinates, size);
		if (error != hipSuccess)
		{
			printf("ERROR in AllocateDeviceSamplePoints: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		error = hipMalloc((void**)&dPoints[i].yCoordinates, size);
		if (error != hipSuccess)
		{
			printf("ERROR in AllocateDeviceSamplePoints: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		error = hipMalloc((void**)&dPoints[i].weights, size);
		if (error != hipSuccess)
		{
			printf("ERROR in AllocateDeviceSamplePoints: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		// By Guiming @ 2016-09-02
		error = hipMalloc((void**)&dPoints[i].distances, size);
		if (error != hipSuccess)
		{
			printf("ERROR in AllocateDeviceSamplePoints: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
	}
	hipSetDevice(0); //Reset device to first GPU
}

//original
//void CopyToDeviceSamplePoints(SamplePoints* dPoints, const SamplePoints hPoints) {
//	int size = hPoints.numberOfPoints * sizeof(float);
//
//	//for(int i = 0; i < hPoints.numberOfPoints; i++)
//	//	printf("x:%.2f y:%.2f w:%.2f\n", hPoints.xCoordinates[i], hPoints.yCoordinates[i], hPoints.weights[i]);
//
//	//printf("copy %d points to device\n", size);
//	hipError_t error;
//
//	error = hipMemcpy(dPoints[0].xCoordinates, hPoints.xCoordinates, size, hipMemcpyHostToDevice);
//	if (error != hipSuccess)
//	{
//		printf("ERROR in CopyToDeviceSamplePoints: %s\n", hipGetErrorString(error));
//		exit(EXIT_FAILURE);
//	}
//	error = hipMemcpy(dPoints[0].yCoordinates, hPoints.yCoordinates, size, hipMemcpyHostToDevice);
//	if (error != hipSuccess)
//	{
//		printf("ERROR in CopyToDeviceSamplePoints: %s\n", hipGetErrorString(error));
//		exit(EXIT_FAILURE);
//	}
//	error = hipMemcpy(dPoints[0].weights, hPoints.weights, size, hipMemcpyHostToDevice);
//	if (error != hipSuccess)
//	{
//		printf("ERROR in CopyToDeviceSamplePoints: %s\n", hipGetErrorString(error));
//		exit(EXIT_FAILURE);
//	}
//
//	// By Guiming @ 2016-09-02
//	error = hipMemcpy(dPoints[0].distances, hPoints.distances, size, hipMemcpyHostToDevice);
//	if (error != hipSuccess)
//	{
//		printf("ERROR in CopyToDeviceSamplePoints: %s\n", hipGetErrorString(error));
//		exit(EXIT_FAILURE);
//	}
//}


//EDIT: Timothy @ 03/26/2021
//Added additional variable to track division of points across multiple GPUs
void CopyToDeviceSamplePoints(SamplePoints* dPoints, const SamplePoints hPoints) {
	int size = hPoints.numberOfPoints * sizeof(float);
	int n = hPoints.numberOfPoints; //Number of points on GPU
	int rem = n % GPU_N; //Remainder to determine if number of GPUs divides Number of Points evenly
	int div = n / GPU_N; //Division of points to be divided amongst GPUs
	int divNum = 0; //Tracks our place in the original n number of points
	int index = 0; //Tracks indexing for our multiple GPUs
	hipError_t error;
	dPoints[0].end = div;

	//Timothy @ 01/15/2020
	//Copying the points to each GPU so the data is present across all devices
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i); //Set device (GPU) being actively copied to
		dPoints[i].start = index; //Begin tracking division of points
		dPoints[i].end = index + div; //Tracking end of division

		//If on last GPU, check if GPU_N divided into points evenly (rem==0) 
		//if not add remainder to size on final GPU
		if ((i == GPU_N - 1) && (rem != 0)) 
		{
			div += rem;
		}
		dPoints[i].numberOfPoints = div; //# of points is assigned here to compensate for remainders.
		error = hipMemcpy(dPoints[i].xCoordinates, hPoints.xCoordinates, size, hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			printf("ERROR 1 in CopyToDeviceSamplePoints: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		error = hipMemcpy(dPoints[i].yCoordinates, hPoints.yCoordinates, size, hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			printf("ERROR 2 in CopyToDeviceSamplePoints: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		error = hipMemcpy(dPoints[i].weights, hPoints.weights, size, hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			printf("ERROR 3 in CopyToDeviceSamplePoints: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		error = hipMemcpy(dPoints[i].distances, hPoints.distances, size, hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			printf("ERROR 4 in CopyToDeviceSamplePoints: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		index = div; //Set starting index of next group of sample points to the end of previous group.
	}
	hipSetDevice(0); //Reset device to first GPU
}

void CopyFromDeviceSamplePoints(SamplePoints hPoints, const SamplePoints* dPoints){
	int size = dPoints[0].numberOfPoints * sizeof(float);
	hipError_t error;

	hipSetDevice(0);

	error = hipMemcpy(hPoints.xCoordinates, dPoints[0].xCoordinates, size, hipMemcpyDeviceToHost);
	if (error != hipSuccess)
    {
        printf("ERROR 1 in CopyFromDeviceSamplePoints: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	error = hipMemcpy(hPoints.yCoordinates, dPoints[0].yCoordinates, size, hipMemcpyDeviceToHost);
		if (error != hipSuccess)
    {
        printf("ERROR 2 in CopyToDeviceSamplePoints: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	error = hipMemcpy(hPoints.weights, dPoints[0].weights, size, hipMemcpyDeviceToHost);
	if (error != hipSuccess)
    {
        printf("ERROR 3 in CopyToDeviceSamplePoints: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	// By Guiming @ 2016-09-02
	error = hipMemcpy(hPoints.distances, dPoints[0].distances, size, hipMemcpyDeviceToHost);
	if (error != hipSuccess)
	{
	    printf("ERROR 4 in CopyToDeviceSamplePoints: %s\n", hipGetErrorString(error));
	    exit(EXIT_FAILURE);
	}
}

// write to .csv file
void WriteSamplePoints(SamplePoints* Points, const char * csvFile){
	FILE *f = fopen(csvFile, "w");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	fprintf(f, "x, y\n");
	for (int p = 0; p < Points->numberOfPoints; p++){
		fprintf(f, "%f, %f\n", Points->xCoordinates[p], Points->yCoordinates[p]);
	}
	fclose(f);
}

// write to .csv file
void WriteSamplePoints(SamplePoints* Points, float* Hs, float* Ws, const char * csvFile){
	FILE *f = fopen(csvFile, "w");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	fprintf(f, "x, y, h, w\n");
	for (int p = 0; p < Points->numberOfPoints; p++){
		fprintf(f, "%f, %f, %f, %f\n", Points->xCoordinates[p], Points->yCoordinates[p], Hs[p], Ws[p]);
	}
	fclose(f);
}

void FreeSamplePoints(SamplePoints* Points) {
	free(Points->xCoordinates);
	Points->xCoordinates = NULL;

	free(Points->yCoordinates);
	Points->yCoordinates = NULL;

	free(Points->weights);
	Points->weights = NULL;
	
	// By Guiming @ 2016-09-02
	free(Points->distances);
	Points->distances = NULL;
}

void FreeDeviceSamplePoints(SamplePoints* dPoints){
	hipError_t error;
	//Timothy @ 10/16/2020
	//Free Memory Across All Devices
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		error = hipFree(dPoints[i].xCoordinates);
		if (error != hipSuccess)
		{
			printf("ERROR 1 in FreeDeviceSamplePoints: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		dPoints->xCoordinates = NULL;

		error = hipFree(dPoints[i].yCoordinates);
		if (error != hipSuccess)
		{
			printf("ERROR 2 in FreeDeviceSamplePoints: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		dPoints->yCoordinates = NULL;

		error = hipFree(dPoints[i].weights);
		if (error != hipSuccess)
		{
			printf("ERROR 3 in FreeDeviceSamplePoints: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		dPoints->weights = NULL;

		// By Guiming @ 2016-09-02
		error = hipFree(dPoints[i].distances);
		if (error != hipSuccess)
		{
			printf("ERROR in FreeDeviceSamplePoints: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		dPoints->distances = NULL;
	}
	hipSetDevice(0); //Reset device to first GPU
}

// this is a mask
AsciiRaster AllocateAsciiRaster(int nCols, int nRows, float xLLCorner, float yLLCorner, float cellSize, float noDataValue){
	AsciiRaster Ascii;

	Ascii.nCols = nCols;
	Ascii.nRows = nRows;
	Ascii.xLLCorner = xLLCorner;
	Ascii.yLLCorner = yLLCorner;
	Ascii.cellSize = cellSize;
	Ascii.noDataValue = noDataValue;

	int size = Ascii.nCols * Ascii.nRows;
	Ascii.elements = (float*)malloc(size * sizeof(float));

	for (int row = 0; row < Ascii.nRows; row++){
		for (int col = 0; col < Ascii.nCols; col++){
			//if (row < 2 || col < 2)
			//	Ascii.elements[row * nCols + col] = Ascii.noDataValue;
			//else
				Ascii.elements[row * nCols + col] = 0.0f;
		}
	}

	return Ascii;
}

// copy a ascii raster
AsciiRaster CopyAsciiRaster(const AsciiRaster anotherAscii){
	AsciiRaster Ascii;

	Ascii.nCols = anotherAscii.nCols;
	Ascii.nRows = anotherAscii.nRows;
	Ascii.xLLCorner = anotherAscii.xLLCorner;
	Ascii.yLLCorner = anotherAscii.yLLCorner;
	Ascii.cellSize = anotherAscii.cellSize;
	Ascii.noDataValue = anotherAscii.noDataValue;

	int size = Ascii.nCols * Ascii.nRows;
	Ascii.elements = (float*)malloc(size * sizeof(float));

	for (int row = 0; row < Ascii.nRows; row++){
		for (int col = 0; col < Ascii.nCols; col++){
			Ascii.elements[row * Ascii.nCols + col] = anotherAscii.elements[row * Ascii.nCols + col];
		}
	}

	return Ascii;
}

// ascii raster read from a .asc file
AsciiRaster ReadAsciiRaster(char * asciiFile){
	FILE *f = fopen(asciiFile, "rt");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	AsciiRaster Ascii;

	const int HEADER_LINE_LENGTH = 64;
	char hdrLine[HEADER_LINE_LENGTH];
	char* pch;
	float meta[6] = {0.f, 0.f, 0.f, 0.f, 0.f, 0.f};

	// read headers
	for (int i = 0; i < 6; i++){
		fgets(hdrLine, HEADER_LINE_LENGTH, f);
		pch = strtok(hdrLine, " \n");
		while (pch != NULL)
		{
			pch = strtok(NULL, "\n");
			meta[i] = atof(pch);
			break;
		}
	}

	Ascii.nCols = (int)meta[0];
	Ascii.nRows = (int)meta[1];
	Ascii.xLLCorner = meta[2];
	Ascii.yLLCorner = meta[3];
	Ascii.cellSize = meta[4];
	Ascii.noDataValue = meta[5];
	Ascii.elements = (float*)malloc(Ascii.nRows * Ascii.nCols * sizeof(float));

	const int DATA_LINE_LENGTH = Ascii.nCols * 32;
	char* datLine = (char*)malloc(DATA_LINE_LENGTH * sizeof(char));

	int row_counter = 0;
	while (fgets(datLine, DATA_LINE_LENGTH, f) != NULL){
		int col_counter = 0;
		pch = strtok(datLine, " \n");
		Ascii.elements[row_counter*Ascii.nCols+col_counter] = atof(pch);
		while (pch != NULL)
		{
			pch = strtok(NULL, " ");
			if (pch != NULL && col_counter < Ascii.nCols - 1){
				col_counter++;
				Ascii.elements[row_counter*Ascii.nCols + col_counter] = atof(pch);
			}
		}
		row_counter++;
	}
	free(datLine);

	fclose(f);

	return Ascii;
}

void AllocateDeviceAsciiRaster(AsciiRaster* dAscii, const AsciiRaster hAscii){
	//Timothy @ 10/16/2020
	//Allocate Memory Across All Devices
	for (int i = 0; i < GPU_N; i++)
	{
		dAscii[i].nCols = hAscii.nCols;
		dAscii[i].nRows = hAscii.nRows;
		dAscii[i].xLLCorner = hAscii.xLLCorner;
		dAscii[i].yLLCorner = hAscii.yLLCorner;
		dAscii[i].cellSize = hAscii.cellSize;
		dAscii[i].noDataValue = hAscii.noDataValue;
		int size = hAscii.nCols*hAscii.nRows * sizeof(float);
		hipError_t error;
	
			hipSetDevice(i);
			error = hipMalloc((void**)&dAscii[i].elements, size);
			if (error != hipSuccess)
			{
				printf("ERROR in AllocateDeviceAsciiRaster: %s\n", hipGetErrorString(error));
				exit(EXIT_FAILURE);
			}
	}
	hipSetDevice(0); //Reset device to first GPU
}

void CopyToDeviceAsciiRaster(AsciiRaster* dAscii, const AsciiRaster hAscii){
	int size = hAscii.nCols*hAscii.nRows * sizeof(float);
	hipError_t error;
	//Copy raster to all available devices
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		error = hipMemcpy(dAscii[i].elements, hAscii.elements, size, hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			printf("ERROR in CopyToDeviceAsciiRaster: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
	}
	hipSetDevice(0); //Reset device to first GPU
}

void CopyFromDeviceAsciiRaster(AsciiRaster hAscii, const AsciiRaster dAscii){
	hAscii.nCols = dAscii.nCols;
	hAscii.nRows = dAscii.nRows;
	hAscii.xLLCorner = dAscii.xLLCorner;
	hAscii.yLLCorner = dAscii.yLLCorner;
	hAscii.cellSize = dAscii.cellSize;
	hAscii.noDataValue = dAscii.noDataValue;

	int size = dAscii.nCols*dAscii.nRows * sizeof(float);
	hipError_t error;
	error = hipMemcpy(hAscii.elements, dAscii.elements, size, hipMemcpyDeviceToHost);
	if (error != hipSuccess)
    {
        printf("ERROR in CopyFromDeviceAsciiRaster: %s\n", hipGetErrorString(error));
		printf("size=%d mode=%d\n", size, hipMemcpyDeviceToHost);
        exit(EXIT_FAILURE);
    }
}

// write to .asc file
void WriteAsciiRaster(AsciiRaster* Ascii, const char * asciiFile){
	FILE *f = fopen(asciiFile, "w");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	fprintf(f, "ncols %d\n", Ascii->nCols);
	fprintf(f, "nrows %d\n", Ascii->nRows);
	fprintf(f, "xllcorner %f\n", Ascii->xLLCorner);
	fprintf(f, "yllcorner %f\n", Ascii->yLLCorner);
	fprintf(f, "cellsize %f\n", Ascii->cellSize);
	fprintf(f, "NODATA_value %.0f\n", Ascii->noDataValue);

	for (int row = 0; row < Ascii->nRows; row++){
		for (int col = 0; col < Ascii->nCols; col++){
			fprintf(f, "%.16f ", Ascii->elements[row*Ascii->nCols+col]);
		}
		fprintf(f, "\n");
	}
	fclose(f);
}

void FreeAsciiRaster(AsciiRaster* Ascii){
	free(Ascii->elements);
	Ascii->elements = NULL;
}

void FreeDeviceAsciiRaster(AsciiRaster* Ascii){
	hipError_t error;
	//Timothy @ 10/16/2020
	//Free Memory Across All Devices
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		error = hipFree(Ascii[i].elements);
		if (error != hipSuccess)
		{
			printf("ERROR in FreeDeviceAsciiRaster: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		Ascii[i].elements = NULL;
	}
	hipSetDevice(0); //Reset device to first GPU
}

// edge effects correction weights at each point, weights is allocated somewhere else
void EdgeCorrectionWeightsExact(SamplePoints Points, float h, AsciiRaster Ascii, float *weights){
	float h2 = h * h;
	float cellArea = Ascii.cellSize * Ascii.cellSize;
	float p_x, p_y, cell_x, cell_y;
	float ew;

	for (int p = 0; p < Points.numberOfPoints; p++){
		//printf("%6d / %6d\n", p, Points.numberOfPoints);

		// By Guiming @ 2016-09-03
		if(Points.distances[p] >= CUT_OFF_FACTOR * h2){ // pnts too far away from the study area boundary, skip to save labor!
			weights[p] = 1.0f;
			//printf("bypassed! %f %f %d\n", Points.distances[p], 9.0 * h2, nThreads);
			continue;
		}

		p_x = Points.xCoordinates[p];
		p_y = Points.yCoordinates[p];
		ew = 0.0f;

		// added by Guiming @2016-09-11
		// narrow down the row/col range
		int row_lower = 0;
		int row_upper = Ascii.nRows - 1;
		int col_lower = 0;
		int col_upper = Ascii.nCols - 1;
		if(NARROW){
			int r = YCOORD_TO_ROW(p_y + SQRT_CUT_OFF_FACTOR * h, Ascii.nRows, Ascii.yLLCorner, Ascii.cellSize);
			row_lower = MAX(0, r);
			row_upper = MIN(Ascii.nRows - 1, YCOORD_TO_ROW(p_y - SQRT_CUT_OFF_FACTOR * h, Ascii.nRows, Ascii.yLLCorner, Ascii.cellSize));
			col_lower = MAX(0, XCOORD_TO_COL(p_x - SQRT_CUT_OFF_FACTOR * h, Ascii.xLLCorner, Ascii.cellSize));
			col_upper = MIN(Ascii.nCols - 1, XCOORD_TO_COL(p_x + SQRT_CUT_OFF_FACTOR * h, Ascii.xLLCorner, Ascii.cellSize));
		}

		for (int row = row_lower; row <= row_upper; row++){
			for (int col = col_lower; col <= col_upper; col++){
				if (Ascii.elements[row*Ascii.nCols+col] != Ascii.noDataValue){
					cell_x = COL_TO_XCOORD(col, Ascii.xLLCorner, Ascii.cellSize);
					cell_y = ROW_TO_YCOORD(row, Ascii.nRows, Ascii.yLLCorner, Ascii.cellSize);
					float d2 = Distance2(p_x, p_y, cell_x, cell_y);
					ew += GaussianKernel(h2, d2) * cellArea;
				}
			}
		}
		weights[p] = 1.0 / ew;
	}
}

void EdgeCorrectionWeightsExact(SamplePoints Points, float* hs, AsciiRaster Ascii, float *weights){
	//float h2 = BandWidth2(Points);
	float cellArea = Ascii.cellSize * Ascii.cellSize;
	float p_x, p_y, cell_x, cell_y;
	float ew, h2;

	for (int p = 0; p < Points.numberOfPoints; p++){
		//printf("%6d / %6d\n", p, Points.numberOfPoints);
		p_x = Points.xCoordinates[p];
		p_y = Points.yCoordinates[p];
		ew = 0.0f;
		h2 = hs[p] * hs[p];

		// By Guiming @ 2016-09-03
		if(Points.distances[p] >= CUT_OFF_FACTOR * h2){ // pnts too far away from the study area boundary, skip to save labor!
			weights[p] = 1.0f;
			//printf("bypassed! %f %f %d\n", Points.distances[p], 9.0 * h2, nThreads);
			continue;
		}

		// added by Guiming @2016-09-11
		// narrow down the row/col range
		int row_lower = 0;
		int row_upper = Ascii.nRows - 1;
		int col_lower = 0;
		int col_upper = Ascii.nCols - 1;

		if(NARROW){
			int r = YCOORD_TO_ROW(p_y + SQRT_CUT_OFF_FACTOR * hs[p], Ascii.nRows, Ascii.yLLCorner, Ascii.cellSize);
			row_lower = MAX(0, r);
			row_upper = MIN(Ascii.nRows - 1, YCOORD_TO_ROW(p_y - SQRT_CUT_OFF_FACTOR * hs[p], Ascii.nRows, Ascii.yLLCorner, Ascii.cellSize));
			col_lower = MAX(0, XCOORD_TO_COL(p_x - SQRT_CUT_OFF_FACTOR * hs[p], Ascii.xLLCorner, Ascii.cellSize));
			col_upper = MIN(Ascii.nCols - 1, XCOORD_TO_COL(p_x + SQRT_CUT_OFF_FACTOR * hs[p], Ascii.xLLCorner, Ascii.cellSize));
		}

		for (int row = row_lower; row <= row_upper; row++){
			for (int col = col_lower; col <= col_upper; col++){
				if (Ascii.elements[row*Ascii.nCols+col] != Ascii.noDataValue){
					cell_x = COL_TO_XCOORD(col, Ascii.xLLCorner, Ascii.cellSize);
					cell_y = ROW_TO_YCOORD(row, Ascii.nRows, Ascii.yLLCorner, Ascii.cellSize);
					float d2 = Distance2(p_x, p_y, cell_x, cell_y);
					ew += GaussianKernel(h2, d2) * cellArea;
				}
			}
		}
		weights[p] = 1.0 / ew;
	}
}

float* AllocateEdgeCorrectionWeights(SamplePoints Points){
	return (float*)malloc(Points.numberOfPoints*sizeof(float));
}

void AllocateDeviceEdgeCorrectionWeights(float** dWeights, SamplePoints Points){
	hipError_t error;
	//Timothy @ 10/16/2020
	//Allocate Memory Across All Devices
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		error = hipMalloc((void**)&dWeights[i], Points.numberOfPoints * sizeof(float));
		if (error != hipSuccess)
		{
			printf("ERROR in AllocateDeviceEdgeCorrectionWeights: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
	}
	hipSetDevice(0); //Reset device to first GPU
}

void FreeEdgeCorrectionWeights(float* weights){
	
	free(weights);
	weights = NULL;
}

void FreeDeviceEdgeCorrectionWeights(float** weights){
	hipError_t error;
	//Timothy @ 10/16/2020
	//Free Memory Across All Devices
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		error = hipFree(weights[i]);
		if (error != hipSuccess)
		{
			printf("ERROR in FreeDeviceEdgeCorrectionWeights: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		weights[i] = NULL;
	}
	hipSetDevice(0); //Reset device to first GPU
}

///////// Guiming on 2016-03-16 ///////////////
// the array holding bandwidth at each point
float* AllocateBandwidths(int n){ // n is number of points
	return (float*)malloc(n*sizeof(float));
}

void AllocateDeviceBandwidths(float** dBandwidths, int n){ // n is number of points
	hipError_t error;
	//Allocate bandwidth accross all available devices
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		error = hipMalloc((void**)&dBandwidths[i], n * sizeof(float));
		if (error != hipSuccess)
		{
			printf("ERROR in AllocateDeviceBandwidths: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
	}
	hipSetDevice(0); //Reset device to first GPU
}

void CopyToDeviceBandwidths(float** dBandwidth, const float* hBandwidths, const int n) {
	int size = n * sizeof(float);
	hipError_t error;
	
	//Copy to each available device
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		error = hipMemcpy(dBandwidth[i], hBandwidths, size, hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			printf("ERROR in CopyToDeviceBandwidths: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
	}
	hipSetDevice(0);
}

void CopyFromDeviceBandwidths(float* hBandwidth, const float* dBandwidths, const int n){
	int size = n * sizeof(float);
	hipError_t error;
	error = hipMemcpy(hBandwidth, dBandwidths, size, hipMemcpyDeviceToHost);
	if (error != hipSuccess)
    {
        printf("ERROR in CopyFromDeviceBandwidths: %s\n", hipGetErrorString(error));
		printf("size=%d mode=%d\n", size, hipMemcpyDeviceToHost);
        exit(EXIT_FAILURE);
    }
}

void FreeDeviceBandwidths(float* bandwidths){
	hipError_t error;
	//Timothy @ 10/16/2020
	//Free Memory Across All Devices
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		error = hipFree(bandwidths);
		if (error != hipSuccess)
		{
			printf("ERROR in FreeDeviceBandwidths: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		bandwidths = NULL;
	}
	hipSetDevice(0); //Reset device to first GPU
}

void FreeBandwidths(float* bandwidths){
	free(bandwidths);
	bandwidths = NULL;
}

// the array holding inclusive density at each point
float* AllocateDen(int n){ // n is number of points
	return (float*)malloc(n*sizeof(float));
}

void AllocateDeviceDen(float** dDen, int n){ // n is number of points
	hipError_t error;
	//Timothy @ 10/16/2020
	//Allocate Memory Across All Devices
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		error = hipMalloc((void**)&dDen[i], n * sizeof(float));
		if (error != hipSuccess)
		{
			printf("ERROR in AllocateDeviceDen: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
	}
	hipSetDevice(0); //Reset device to first GPU
}

void CopyToDeviceDen(float** dDen, const float* hDen, const int n){
	int size = n * sizeof(float);
	hipError_t error;
	//Copy accross all available devices
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		error = hipMemcpy(dDen[i], hDen, size, hipMemcpyHostToDevice);
	}
	if (error != hipSuccess)
    {
        printf("ERROR in CopyToDeviceDen: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	hipSetDevice(0); //Reset device to first GPU
}

void CopyFromDeviceDen(float* hDen, const float* dDen, const int n){
	int size = n * sizeof(float);
	hipError_t error;
	error = hipMemcpy(hDen, dDen, size, hipMemcpyDeviceToHost);
	if (error != hipSuccess)
    {
        printf("ERROR in CopyFromDeviceDen: %s\n", hipGetErrorString(error));
		printf("size=%d mode=%d\n", size, hipMemcpyDeviceToHost);
        exit(EXIT_FAILURE);
    }
}

void CopyDeviceDen(float* dDenTo, const float* dDenFrom, const int n){
	int size = n * sizeof(float);
	hipError_t error = hipSuccess;
	error = hipMemcpy(dDenTo, dDenFrom, size, hipMemcpyDeviceToDevice);
	if (error != hipSuccess)
    {
        printf("ERROR in CopyDeviceDen: %s\n", hipGetErrorString(error));
		printf("size=%d mode=%d\n", size, hipMemcpyDeviceToDevice);
        exit(EXIT_FAILURE);
    }
}

void FreeDeviceDen(float** den){
	hipError_t error;
	hipSetDevice(0);
	error = hipFree(den[0]);
	if (error != hipSuccess)
	{
		printf("ERROR in FreeDeviceDen(Elements): %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	den = NULL;
	error = hipFree(den);
	if (error != hipSuccess)
	{
		printf("ERROR in FreeDeviceDen(Pointer): %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void FreeDen(float* den){
	free(den);
	den = NULL;
}

// compute the optimal Maximum Likelihood Estimation fixed bandwidth
// By Guiming @ 2016-02-26
float MLE_FixedBandWidth(AsciiRaster* Ascii, SamplePoints* Points, float **edgeWeights, float h, float* den0, float* den1, bool useGPU, float** dDen0, float** dDen1){
	
	float hA = h/10;
	float hD = 4 * h;
	float width = hD - hA;
	float epsilon = width/100;
	float factor = 1 + sqrtf(5.0f);
	int iteration = 0;

	printf("hA: %f hD: %f width: %f, epsilon: %d\n", hA, hD, width, epsilon); //DEBUG
	while(width > epsilon){

		if(DEBUG){
			printf("iteration: %d ", iteration);
			printf("hD: %.6f ", hD);
			printf("hA: %.6f ", hA);
		}

		float hB = hA + width / factor;
		float hC = hD - width / factor;

		//ERROR HERE, ONLY WHEN USING GPU
		float LoghB = LogLikelihood(Ascii, Points, edgeWeights, hB, den0, den1, useGPU, dDen0, dDen1);
		float LoghC = LogLikelihood(Ascii, Points, edgeWeights, hC, den0, den1, useGPU, dDen0, dDen1);

		if(LoghB > LoghC){
			hD = hC;
			if(DEBUG) printf("LoghB: %.6f \n", LoghB);
		}
		else{
			hA = hB;
			if(DEBUG) printf("LoghC: %.6f \n", LoghC);
		}

		width = hD - hA;

		iteration += 1;
	}

	return (hA + hD) / 2;
}

// By Guiming @ 2016-05-21
// computed fixed bandwidth kde
void ComputeFixedDensityAtPoints(AsciiRaster Ascii, SamplePoints Points, float* edgeWeights, float h, float* den0, float* den1, float* dDen0, float* dDen1) {
	
	int numPoints = Points.numberOfPoints;
		// update edge correction weights
		if (UPDATEWEIGHTS) {
			EdgeCorrectionWeightsExact(Points, h, Ascii, edgeWeights);
		}

		for (int i = 0; i < numPoints; i++) {
			float pi_x = Points.xCoordinates[i];
			float pi_y = Points.yCoordinates[i];

			float den = EPSILONDENSITY;
			float den_itself = EPSILONDENSITY;
			for (int j = 0; j < numPoints; j++) {
				float pj_x = Points.xCoordinates[j];
				float pj_y = Points.yCoordinates[j];
				float pj_w = Points.weights[j];
				float pj_ew = edgeWeights[j];

				float d2 = Distance2(pi_x, pi_y, pj_x, pj_y);

				if (j == i) {
					den_itself += GaussianKernel(h * h, d2) * pj_w * pj_ew; // / numPoints;
				}
				else {
					den += GaussianKernel(h * h, d2) * pj_w * pj_ew;
				}
			}

			if (den0 != NULL) {
				den0[i] = den + den_itself;
			}
			if (den1 != NULL) {
				den1[i] = den;
			}
		}
}

// By Timothy @ 04-23-2021
// Separated this function into two separate, rather than using the boolean value as was done previously.
//this was done to enable parallel functionality accross multiple GPUs
void ComputeFixedDensityDevice(hipStream_t* streams, AsciiRaster* Ascii, SamplePoints* Points, float** edgeWeights, float h, float* den0, float* den1, float** dDen0, float** dDen1){
	int numPoints = Points[0].numberOfPoints;
	// invoke kernels to compute density at each point
	// execution config.
	int NBLOCK_W = (numPoints + BLOCK_SIZE - 1) / BLOCK_SIZE;
	int GRID_SIZE_W = (int)(sqrtf(NBLOCK_W)) + 1;
	dim3 dimGrid_W(GRID_SIZE_W, GRID_SIZE_W);
	for (int i = 0; i < GPU_N; i++){
		hipSetDevice(i);
		// update edge correction weights
		if (UPDATEWEIGHTS) {
			CalcEdgeCorrectionWeights <<<dimGrid_W, BLOCK_SIZE, 0, streams[i]>>>(h * h, Points[i], Ascii[i], edgeWeights[i]);
		}
		//// brute force to search for neighbors
		//DensityAtPoints<<<dimGrid_W, BLOCK_SIZE>>>(h*h, Points, edgeWeights, dDen0, dDen1);
		///// use KD Tree to speedup neighor search
		//CopyToDeviceDen(gpuDen, zeroDen, Points.numberOfPoints);
		DensityAtPointsKdtr <<<dimGrid_W, BLOCK_SIZE, 0, streams[i]>>>(GPU_tree[i].m_gpu_nodes, GPU_tree[i].m_gpu_indexes, GPU_tree[0].m_gpu_points, h * h, Points[i], edgeWeights[i], gpuDen[i]);
		
		// have to do this as a separate kernel call due to the need of block synchronization !!!
		// this took me hours to debug!
		dCopyDensityValues <<<dimGrid_W, BLOCK_SIZE, 0, streams[i]>>>(Points[i], edgeWeights[i], h * h, gpuDen[i], dDen0[i], dDen1[i]);
		hipStreamSynchronize(streams[i]);
	}
	hipSetDevice(0);
}

// By Guiming @ 2016-02-26
// the log likelihood given single bandwidth h
float LogLikelihood(AsciiRaster* Ascii, SamplePoints* Points, float **edgeWeights, float h, float* den0, float* den1, bool useGPU, float** dDen0, float** dDen1){
	int numPoints = Points[0].numberOfPoints;
	float logL = 0.0f; // log likelihood
	int size = Points[0].totalNumPoints * sizeof(float);
	hipError_t error = hipSuccess;

	if (error != hipSuccess)
	{
		printf("ERROR 0 in LogLikelihood: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	if (useGPU) { // do it on GPU
		//SamplePoints hostP = AllocateSamplePoints(Points[0].numberOfPoints);
		for (int i = 0; i < GPU_N; i++)
		{
			hipSetDevice(i);
			///*
			// execution config.
			int NBLOCK_W = (numPoints + BLOCK_SIZE - 1) / BLOCK_SIZE;
			int GRID_SIZE_W = (int)(sqrtf(NBLOCK_W)) + 1;
			dim3 dimGrid_W(GRID_SIZE_W, GRID_SIZE_W);

			// update edge correction weights
			if (UPDATEWEIGHTS) {
				CalcEdgeCorrectionWeights << <dimGrid_W, BLOCK_SIZE, 0, streams[i] >> > (h * h, Points[i], Ascii[i], edgeWeights[i]);
			}
			hipStreamSynchronize(streams[i]);
			DensityAtPointsKdtr << <dimGrid_W, BLOCK_SIZE, 0, streams[i] >> > (GPU_tree[i].m_gpu_nodes, GPU_tree[i].m_gpu_indexes, GPU_tree[i].m_gpu_points, h * h, Points[i], edgeWeights[i], gpuDen[i]);
			// have to do this as a separate kernel call due to the need of block synchronization !!!
			// this took me hours to debug!
			hipStreamSynchronize(streams[i]);
			dCopyDensityValues << <dimGrid_W, BLOCK_SIZE, 0, streams[i] >> > (Points[i], edgeWeights[i], h * h, gpuDen[i], NULL, dDen1[i]);
			hipStreamSynchronize(streams[i]);
		}
		ReformPoints(Points);
		// compute likelihood on GPU
		ReductionSumGPU(dDen1[0], numPoints);
		hipMemcpyFromSymbol(&logL, HIP_SYMBOL(dReductionSum), sizeof(float), 0, hipMemcpyDeviceToHost);
		//printf("reduction result (likelihood) A: %3.4f \n", logL);
		CopyToDeviceSamplePoints(Points, sPoints);
		//Cleanup
		hipSetDevice(0);
		/*FreeSamplePoints(&hostP);*/
	}
	else{ // do it on CPU
		// update edge correction weights
		if(UPDATEWEIGHTS){
			EdgeCorrectionWeightsExact(Points[0], h, Ascii[0], edgeWeights[0]);
		}

		// the kd tree appraoch
		float* tmpden = AllocateDen(numPoints);
		float h2 = h * h;
		float range = CUT_OFF_FACTOR * h2;

		for(int i = 0; i < numPoints; i++){
			tmpden[i] = -1.0 * GaussianKernel(h2, 0.0f) *  Points[0].weights[i] * edgeWeights[0][i];
		}

		vector<int> ret_index = vector<int>();
		vector<float> ret_dist = vector<float>(); // squared distance

		for(int i = 0; i < numPoints; i++){
			float pi_x = Points[0].xCoordinates[i];
			float pi_y = Points[0].yCoordinates[i];
			float pj_w = Points[0].weights[i];
			float pj_ew = edgeWeights[0][i];

			// range query
			Point query;
			query.coords[0] = pi_x;
			query.coords[1] = pi_y;
			ret_index.clear();
			ret_dist.clear();
			tree.SearchRange(query, range, ret_index, ret_dist);
			//printf("CPU PNT_%d %d NBRS RANGE=%.1f\n", i, ret_index.size(), range);

			if(ret_index.size() > MAX_N_NBRS) MAX_N_NBRS = ret_index.size();

			float g = 0.0f;
			int idx;
			for(int j = 0; j < ret_index.size(); j++){
					g = GaussianKernel(h2, ret_dist[j]) * pj_w *pj_ew;
					idx = ret_index[j];
					//float t = tmpden[idx];
					tmpden[idx] += g;
					//if(i == 0) printf("CPU PNT_%d g[%d]=%.5f gpuDen[%d]=%.5f gpuDen[%d]=%.5f\n", i, idx, g, idx, t, idx, tmpden[idx]);
			}
		} // END OF COMPUTING DENSITIES AT POINTS



		for(int i = 0; i < numPoints; i++){
			//printf("CPU H2=%.2f DEN[%d]=%.5f\n", h2, i, tmpden[i]);
			logL += logf(tmpden[i] + EPSILONDENSITY);
		}

		if(den0 != NULL){
			for(int i = 0; i < numPoints; i++)
				den0[i] = tmpden[i]  + GaussianKernel(h2, 0.0f) * Points[0].weights[i] * edgeWeights[0][i];
		}
		if(den1 != NULL){
			for(int i = 0; i < numPoints; i++)
				den1[i] = tmpden[i];
		}

		FreeDen(tmpden);
	}
	return logL;
}

// the log likelihood given bandwidths hs
// By Guiming @ 2016-02-26
// float* den0 : density based on all points, including itself
// float* den1 : leave one out density
//EDIT: Timothy @ 03/12/2021
//Added additional variable passed in, so now when useGPU is TRUE, the function will be handled across however many GPUs are available
float LogLikelihood(AsciiRaster* Ascii, SamplePoints* Points, SamplePoints* gpuPoints, float **edgeWeights, float* hs, float* den0, float* den1, bool useGPU, float** dHs, float** dDen0, float** dDen1, float h, float alpha, float** dDen0cpy){
	int numPoints = Points[0].numberOfPoints;
	float logL = 0.0f; // log likelihood
	hipError_t error = hipSuccess;
	// execution config.
	int NBLOCK_W = (numPoints + BLOCK_SIZE - 1) / BLOCK_SIZE;
	int GRID_SIZE_W = (int)(sqrtf(NBLOCK_W)) + 1;
	dim3 dimGrid_W(GRID_SIZE_W, GRID_SIZE_W);
	if(useGPU){ // do it on GPU
		for (int i = 0; i < GPU_N; i++)
		{
			hipSetDevice(i);
			//CopyToDeviceBandwidths(dHs, hs, numPoints);

			// update bandwidth on GPU
			//CalcVaryingBandwidths<<<dimGrid_W, BLOCK_SIZE>>>(Points, dDen0, h, alpha, dHs);
			// update edge correction weights
			if (UPDATEWEIGHTS) {
				CalcEdgeCorrectionWeights << <dimGrid_W, BLOCK_SIZE, 0, streams[i] >> > (h * h, gpuPoints[i], Ascii[i], edgeWeights[i]);
			}

			// compute (log) density at sample points [h^2, not h! OMG!!! Took me hours for spotting this!]
			//DensityAtPoints<<<dimGrid_W, BLOCK_SIZE>>>(h * h, Points, edgeWeights, dDen0, dDen1);

			//CopyToDeviceDen(gpuDen, zeroDen, Points.numberOfPoints);
			DensityAtPointsKdtr << <dimGrid_W, BLOCK_SIZE, 0, streams[i] >> > (GPU_tree[i].m_gpu_nodes, GPU_tree[i].m_gpu_indexes, GPU_tree[i].m_gpu_points, h * h, gpuPoints[i], edgeWeights[i], gpuDen[i]);
			// have to do this as a separate kernel call due to the need of block synchronization !!!
			// this took me hours to debug!
			dCopyDensityValues << <dimGrid_W, BLOCK_SIZE, 0, streams[i] >> > (gpuPoints[i], edgeWeights[i], h * h, gpuDen[i], dDen0[i], dDen1[i]);
			
			hipStreamSynchronize(streams[i]);
		}

		ReformDensities(dDen0, den0);
		 

		CopyDeviceDen(dDen0cpy[0], dDen0[0], sPoints.numberOfPoints);
		
		int size = gpuPoints[0].totalNumPoints * sizeof(float);
		//SamplePoints hostP = AllocateSamplePoints(Points[0].numberOfPoints);
		if (error != hipSuccess)
		{
			printf("ERROR 0 in LogLikelihood (Overload): %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		ReformPoints(gpuPoints);
		
		// compute sum of log densities on GPU
		ReductionSumGPU(dDen0cpy[0], gpuPoints[0].totalNumPoints);
		//float tmp = 0.0f;
		//hipMemcpyFromSymbol(&tmp, HIP_SYMBOL(dReductionSum), sizeof(float), 0, hipMemcpyDeviceToHost);
		//printf("reduction result (geometricmean): %3.4f \n", exp(tmp/numPoints));
		
		CopyToDeviceSamplePoints(gpuPoints, sPoints);
		
		for (int i = 0; i < GPU_N; i++)
		{
			hipSetDevice(i);
			// update bandwidth on GPU
			CalcVaryingBandwidths << <dimGrid_W, BLOCK_SIZE, 0, streams[i] >> > (gpuPoints[i], dDen0[i], h, alpha, dHs[i]);

			// update edge correction weights
			if (UPDATEWEIGHTS) {
				CalcEdgeCorrectionWeights << <dimGrid_W, BLOCK_SIZE, 0, streams[i] >> > (dHs[i], gpuPoints[i], Ascii[i], edgeWeights[i]);
			}

			//DensityAtPoints<<<dimGrid_W, BLOCK_SIZE>>>(dHs, Points, edgeWeights, dDen0, dDen1);
			//CopyToDeviceDen(gpuDen, zeroDen, Points.numberOfPoints);

			//to-do: BANDWIDTH AND EDGECORRECTIONWEIGHTS NEEDS TO BE REFORMED HERE! Just like densities

			DensityAtPointsKdtr << <dimGrid_W, BLOCK_SIZE, 0, streams[i] >> > (GPU_tree[i].m_gpu_nodes, GPU_tree[i].m_gpu_indexes, GPU_tree[i].m_gpu_points, dHs[i], gpuPoints[i], edgeWeights[i], gpuDen[i]);
			// have to do this as a separate kernel call due to the need of block synchronization !!!
			// this took me hours to debug!
			dCopyDensityValues << <dimGrid_W, BLOCK_SIZE, 0, streams[i] >> > (gpuPoints[i], edgeWeights[i], dHs[i], gpuDen[i], dDen0[i], dDen1[i]);
		
			hipStreamSynchronize(streams[i]);
		}
		ReformPoints(gpuPoints);
		// compute likelihood on GPU
		ReductionSumGPU(dDen1[0], numPoints);
		hipMemcpyFromSymbol(&logL, HIP_SYMBOL(dReductionSum), sizeof(float), 0, hipMemcpyDeviceToHost);
		CopyToDeviceSamplePoints(gpuPoints, sPoints);
		//printf("reduction result (likelihood): %3.4f \n", logL);
		//Cleanup
		//FreeSamplePoints(&hostP);
		hipSetDevice(0);
	}
	else{ // do it on CPU
		// update edge correction weights
		if(UPDATEWEIGHTS){
			EdgeCorrectionWeightsExact(Points[0], h, Ascii[0], edgeWeights[0]);
		}

		// kdtree approach
		float h2 = h * h;
		float range = CUT_OFF_FACTOR * h2;
		float* denTmp = AllocateDen(numPoints);
		for(int i = 0; i < numPoints; i++){
			denTmp[i] = 0.0f;
		}

		vector<int> ret_index = vector<int>();
		vector<float> ret_dist = vector<float>(); // squared distance
		for(int i = 0; i < numPoints; i++){
			float pi_x = Points[0].xCoordinates[i];
			float pi_y = Points[0].yCoordinates[i];
			float pj_w = Points[0].weights[i];
			float pj_ew = edgeWeights[0][i];

			// range query
			Point query;
			query.coords[0] = pi_x;
			query.coords[1] = pi_y;
			ret_index.clear();
			ret_dist.clear();
			tree.SearchRange(query, range, ret_index, ret_dist);

			if(ret_index.size() > MAX_N_NBRS) MAX_N_NBRS = ret_index.size();

			int nn = ret_index.size();
			float g = 0.0f;
			int idx;
			for(int j = 0; j < ret_index.size(); j++){
					g = GaussianKernel(h2, ret_dist[j]) * pj_w * pj_ew;
					idx = ret_index[j];
					denTmp[idx] += g;
			}
		} // END OF COMPUTING DENSITIES AT POINTS

		// update bandwidths
		//float gml = compGML(den0, numPoints);
		float gml = compGML(denTmp, numPoints);
	    for(int i = 0; i < numPoints; i++){
				hs[i] = h * powf((denTmp[i] / gml), alpha);
	    }

		// update edge correction weights
		if(UPDATEWEIGHTS){
			EdgeCorrectionWeightsExact(Points[0], hs, Ascii[0], edgeWeights[0]);
		}

		for(int i = 0; i < numPoints; i++){
			float h2 = hs[i] * hs[i];
			denTmp[i] = -1.0 * GaussianKernel(h2, 0.0f) *  Points[0].weights[i] * edgeWeights[0][i];
		}

		for(int i = 0; i < numPoints; i++){
			float pi_x = Points[0].xCoordinates[i];
			float pi_y = Points[0].yCoordinates[i];
			float pj_w = Points[0].weights[i];
			float pj_ew = edgeWeights[0][i];
			float h2 = hs[i] * hs[i];
			float range = CUT_OFF_FACTOR * h2;

			// range query
			Point query;
			query.coords[0] = pi_x;
			query.coords[1] = pi_y;
			ret_index.clear();
			ret_dist.clear();
			tree.SearchRange(query, range, ret_index, ret_dist);

			if(ret_index.size() > MAX_N_NBRS) MAX_N_NBRS = ret_index.size();

			int nn = ret_index.size();
			float g = 0.0f;
			int idx;
			for(int j = 0; j < ret_index.size(); j++){
					g = GaussianKernel(h2, ret_dist[j]) * pj_w * pj_ew;
					idx = ret_index[j];
					denTmp[idx] += g;
			}
		} // END OF COMPUTING DENSITIES AT POINTS

		for(int i = 0; i < numPoints; i++){
			logL += logf(denTmp[i] + EPSILONDENSITY);
		}

		if(den0 != NULL){
			for(int i = 0; i < numPoints; i++){
				float h2 = hs[i] * hs[i];
				den0[i] = denTmp[i] + GaussianKernel(h2, 0.0f) *  Points[0].weights[i] * edgeWeights[0][i];
			}
		}

		if(den1 != NULL){
			for(int i = 0; i < numPoints; i++){
				den1[i] = denTmp[i];
			}
		}

		FreeDen(denTmp);
	}

	return logL;
}

// compute the log likelihood given a center (h0, alpha0) and step (stepH, stepA)
// By Guiming @ 2016-03-06
/*
 return 9 elements log likelihood in float* logLs
**/
void hj_likelihood(AsciiRaster* Ascii, SamplePoints* Points, SamplePoints* gpuPoints, float** edgeWeights, float h0, float alpha0, float stepH, float stepA, int lastdmax, float* logLs, float* hs, float* den0, float* den1, bool useGPU, float** dHs, float** dDen0, float** dDen1, float** dDen0cpy)
{
    //int n = Points.numberOfPoints;

    //float gml;
    // the center (h0, alpha0)
    if(lastdmax == -1){ // avoid unnecessary [expensive] computation
	    //LogLikelihood(Ascii, Points, edgeWeights, h0, den0, den1, useGPU, dDen0, dDen1);
	    float L0 = LogLikelihood(Ascii, Points, gpuPoints, edgeWeights, hs, den0, den1, useGPU, dHs, dDen0, dDen1, h0, alpha0, dDen0cpy);
	    //printf("L0: %.5f\t", L0);
	    logLs[0] = L0;
	}
    // (h0 - stepH, alpha0)
    if(lastdmax != 2){ // avoid unnecessary [expensive] computation
	    //LogLikelihood(Ascii, Points, edgeWeights, h0 - stepH, den0, den1, useGPU, dDen0, dDen1);
	    float L1 = LogLikelihood(Ascii, Points, gpuPoints, edgeWeights, hs, den0, den1, useGPU, dHs, dDen0, dDen1, h0 - stepH, alpha0, dDen0cpy);
	    //printf("L1: %.5f\t", L1);
	    logLs[1] = L1;
	}
    // (h0 + stepH, alpha0)
    if(lastdmax != 1){
	    //LogLikelihood(Ascii, Points, edgeWeights, h0 + stepH, den0, den1, useGPU, dDen0, dDen1);
	    float L2 = LogLikelihood(Ascii, Points, gpuPoints, edgeWeights, hs, den0, den1, useGPU, dHs, dDen0, dDen1, h0 + stepH, alpha0, dDen0cpy);
	    //printf("L2: %.5f\t", L2);
	    logLs[2] = L2;
	}
    // (h0, alpha0 + stepA)
    if(lastdmax != 4){
	    //LogLikelihood(Ascii, Points, edgeWeights, h0, den0, den1, useGPU, dDen0, dDen1);
	    float L3 = LogLikelihood(Ascii, Points, gpuPoints, edgeWeights, hs, den0, den1, useGPU, dHs, dDen0, dDen1, h0, alpha0 + stepA, dDen0cpy);
	    //printf("L3: %.5f\t", L3);
	    logLs[3] = L3;
	}
    // (h0, alpha0 - stepA)
    if(lastdmax != 3){
	    //LogLikelihood(Ascii, Points, edgeWeights, h0, den0, den1, useGPU, dDen0, dDen1);
	    float L4 = LogLikelihood(Ascii, Points, gpuPoints, edgeWeights, hs, den0, den1, useGPU, dHs, dDen0, dDen1, h0, alpha0 - stepA, dDen0cpy);
	    //printf("L4: %.5f\n", L4);
	    logLs[4] = L4;
	}
}

// compute the optimal h and alpha (parameters for calculating the optimal adaptive bandwith)
// By Guiming @ 2016-03-06
/*
 return 3 optmal parameters in float* optParas (optH, optAlpha, LogLmax)
//EDIT: Timothy @ 03/10/2021
//Added aditional variable to this, hj_likelihood and LogLikelihood functions to handle array of SamplePoints whenever multiple GPUs are present
**/
void hooke_jeeves(AsciiRaster* Ascii, SamplePoints* Points, SamplePoints* gpuPoints, float **edgeWeights, float h0, float alpha0, float stepH, float stepA, float* optParas, float* hs, float* den0, float* den1, bool useGPU, float** dHs, float** dDen0, float** dDen1, float** dDen0cpy){
	float* Ls = (float*)malloc(5 * sizeof(float)); // remember to free at the end
	hj_likelihood(Ascii, Points, gpuPoints, edgeWeights, h0, alpha0, stepH, stepA, -1, Ls, hs, den0, den1, useGPU, dHs, dDen0, dDen1, dDen0cpy);

	float Lmax = Ls[0];

	float s = stepH / 20;
	float a = stepA / 20;

	int iteration = 0;
    while ((stepH > s || stepA > a) &&  iteration <= MAX_NUM_ITERATIONS){

        //float Lmax0 = Lmax;
        int dmax = 0;
        for(int i = 0; i < 5; i++){
            if(Ls[i] > Lmax){
            	Lmax = Ls[i];
                dmax = i;
            }
        }
        if(DEBUG)
        	printf ("iteration: %d center: (%.5f %.5f) steps: (%.5f %.5f) dmax: %d Lmax: %.5f \n", iteration, h0, alpha0, stepH, stepA, dmax, Lmax);

        if(dmax == 0){
            stepH = stepH / 2;
            stepA = stepA / 2;
        }

        else{
            if(dmax == 1){
                h0 = h0 - stepH;
                alpha0 = alpha0;
                Ls[2] = Ls[0]; // avoid unnecessary [expensive] computation
                Ls[0] = Ls[1];
            }
            if(dmax == 2){
                h0 = h0 + stepH;
                alpha0 = alpha0;
                Ls[1] = Ls[0];
                Ls[0] = Ls[2];
            }
            if (dmax == 3){
                h0 = h0;
                alpha0 = alpha0 + stepA;
                Ls[3] = Ls[0];
                Ls[0] = Ls[4];
            }
            if(dmax == 4){
                h0 = h0;
                alpha0 = alpha0 - stepA;
                Ls[3] = Ls[0];
                Ls[0] = Ls[4];
            }
        }
	    hj_likelihood(Ascii, Points, gpuPoints, edgeWeights, h0, alpha0, stepH, stepA, dmax, Ls, hs, den0, den1, useGPU, dHs, dDen0, dDen1, dDen0cpy);
	    iteration++;
    }

    optParas[0] = h0;
    optParas[1] = alpha0;
    optParas[2] = Lmax;

	free(Ls);
    Ls = NULL;
}

///////// Guiming on 2016-03-16 ///////////////

// check whether the result from sequential computation and that from parallel computation agree
void CheckResults(AsciiRaster AsciiSEQ, AsciiRaster AsciiPARA){
	float eps = 0.000001f;

	int n = AsciiSEQ.nCols * AsciiSEQ.nRows;

	for(int i = 0; i < n; i++){
		if(abs(AsciiSEQ.elements[i] - AsciiPARA.elements[i]) > eps){
			printf("TEST FAILED. Result from parallel computation does not match that from sequential computation.\n");
			return;
		}
	}
	printf("TEST PASSED. Result from GPU computation does match that from CPU computation.\n");
}

float compGML(float* den0, int n){
	float gml = 0.0f;
	for(int i = 0; i < n; i++){
		gml = gml + log(den0[i]);
	}
	gml = expf(gml / n);
	return gml;
}

// reduction sum on GPU
void ReductionSumGPU(float* dArray, int numberOfElements){

   unsigned int N = numberOfElements;

   int iteration = 0;
   int NUM_ACTIVE_ITEMS = numberOfElements; // # active items need to be reduced

   // approx. # of blocks needed
   int NUM_BLOCKS = (numberOfElements ) / BLOCK_SIZE;

   // decide grid dimension
   int GRID_SIZE = (int)(sqrtf(NUM_BLOCKS)) + 1;
   dim3 dimGrid(GRID_SIZE, GRID_SIZE);

   // call the kernel for the first iteration
   ReductionSum<<<dimGrid, BLOCK_SIZE>>>(dArray, N, iteration, NUM_ACTIVE_ITEMS);

   // update # of items to be reduced in next iteration
   NUM_ACTIVE_ITEMS = (NUM_ACTIVE_ITEMS + BLOCK_SIZE - 1) / BLOCK_SIZE;

   // update numberOfElements (needed for deciding grid dimension)
   numberOfElements = dimGrid.x * dimGrid.y;

   // increment iteraton index
   iteration++;

   // iterate if needed
   while(numberOfElements > 1){
      NUM_BLOCKS = (numberOfElements ) / BLOCK_SIZE;

      GRID_SIZE = (int)(sqrtf(NUM_BLOCKS)) + 1;
      dimGrid.x = GRID_SIZE;
      dimGrid.y = GRID_SIZE;
      ReductionSum<<<dimGrid, BLOCK_SIZE>>>(dArray, N, iteration, NUM_ACTIVE_ITEMS);
      NUM_ACTIVE_ITEMS = (NUM_ACTIVE_ITEMS + BLOCK_SIZE - 1) / BLOCK_SIZE;

      numberOfElements = dimGrid.x * dimGrid.y;

      iteration++;
   }
}

// mark the boundary cells on a raster representing the study area
// By Guiming @ 2016-09-02
void MarkBoundary(AsciiRaster Ascii, bool useGPU){

	if(useGPU){ // do it on GPU
			// invoke kernels to mark the boundary of study area
			// execution config.
			int NBLOCK_W = (Ascii.nRows *  Ascii.nCols + BLOCK_SIZE - 1) / BLOCK_SIZE;
			int GRID_SIZE_W = (int)(sqrtf(NBLOCK_W)) + 1;
			dim3 dimGrid_W(GRID_SIZE_W, GRID_SIZE_W);
			//printf("In Marking Boundary...\n");
			dMarkBoundary <<<dimGrid_W, BLOCK_SIZE / 2 >> > (Ascii);
	}
	else{ // do it on CPU
		for(int row = 0; row < Ascii.nRows; row++){
			for(int col = 0; col < Ascii.nCols; col++){

				if(Ascii.elements[row * Ascii.nCols + col] == Ascii.noDataValue)
					continue;

				if(row == 0 || (row == Ascii.nRows - 1) || col == 0 || (col == Ascii.nCols - 1)){ // cells in the outmost rows and cols
					Ascii.elements[row * Ascii.nCols + col] = 1.0f;
					continue;
				}

				if(Ascii.elements[(row - 1) * Ascii.nCols + col - 1] == Ascii.noDataValue){
					Ascii.elements[row * Ascii.nCols + col] = 1.0f;
					continue;
				}
				if(Ascii.elements[row * Ascii.nCols + col - 1] == Ascii.noDataValue){
					Ascii.elements[row * Ascii.nCols + col] = 1.0f;
					continue;
				}
				if(Ascii.elements[(row + 1) * Ascii.nCols + col - 1] == Ascii.noDataValue){
					Ascii.elements[row * Ascii.nCols + col] = 1.0f;
					continue;
				}

				if(Ascii.elements[(row - 1) * Ascii.nCols + col] == Ascii.noDataValue){
					Ascii.elements[row * Ascii.nCols + col] = 1.0f;
					continue;
				}
				if(Ascii.elements[(row + 1) * Ascii.nCols + col] == Ascii.noDataValue){
					Ascii.elements[row * Ascii.nCols + col] = 1.0f;
					continue;
				}

				if(Ascii.elements[(row - 1) * Ascii.nCols + col + 1] == Ascii.noDataValue){
					Ascii.elements[row * Ascii.nCols + col] = 1.0f;
					continue;
				}
				if(Ascii.elements[row * Ascii.nCols + col + 1] == Ascii.noDataValue){
					Ascii.elements[row * Ascii.nCols + col] = 1.0f;
					continue;
				}
				if(Ascii.elements[(row + 1) * Ascii.nCols + col + 1] == Ascii.noDataValue){
					Ascii.elements[row * Ascii.nCols + col] = 1.0f;
					continue;
				}
				Ascii.elements[row * Ascii.nCols + col] = 0.0f;
			}
		}
	}
}

// compute the closest distances from sample points to study area boundary
// By Guiming @ 2016-09-02
void CalcDist2Boundary(SamplePoints Points, AsciiRaster Ascii, bool useGPU){

	// mark the boundary first
	//MarkBoundary(Ascii, useGPU); // either on GPU or CPU

	//printf("Done Marking Boundary!\n");

	if(useGPU){ // do it on GPU
			// invoke kernels to compute the nearest distance to boundary (squared) at each point
			// execution config.
			int NBLOCK_W = (Points.numberOfPoints + BLOCK_SIZE - 1) / BLOCK_SIZE;
	    int GRID_SIZE_W = (int)(sqrtf(NBLOCK_W)) + 1;
	    dim3 dimGrid_W(GRID_SIZE_W, GRID_SIZE_W);

			dCalcDist2Boundary<<<dimGrid_W, BLOCK_SIZE>>>(Points, Ascii);
	}
	else{
		float p_x, p_y, cell_x, cell_y;
		for (int p = 0; p < Points.numberOfPoints; p++){
			float minDist = FLOAT_MAX;
			p_x = Points.xCoordinates[p];
			p_y = Points.yCoordinates[p];

			for (int row = 0; row < Ascii.nRows; row++){
				for (int col = 0; col < Ascii.nCols; col++){
					if (Ascii.elements[row*Ascii.nCols+col] == 1.0f){ // cells on boundary
						cell_x = COL_TO_XCOORD(col, Ascii.xLLCorner, Ascii.cellSize);
						cell_y = ROW_TO_YCOORD(row, Ascii.nRows, Ascii.yLLCorner, Ascii.cellSize);
						float d2 = Distance2(p_x, p_y, cell_x, cell_y);

						if(d2 < minDist){
							minDist = d2;
						}
					}
				}
			}

			Points.distances[p] = minDist;
			//printf("p: %d Points.distances[p]: %f minDist: %f\n", p, Points.distances[p]);
		}
	}
}

// By Guiming @ 2016-09-04
SamplePoints CopySamplePoints(const SamplePoints anotherPoints){ // copy points
	int n = anotherPoints.numberOfPoints;
	SamplePoints Points = AllocateSamplePoints(n);
	Points.numberOfPoints = n;
	Points.totalNumPoints = n;
	for(int p = 0; p < n; p++){
		Points.xCoordinates[p] = anotherPoints.xCoordinates[p];
		Points.yCoordinates[p] = anotherPoints.yCoordinates[p];
		Points.weights[p] = anotherPoints.weights[p];
		Points.distances[p] = anotherPoints.distances[p];
	}
	return Points;
}

// comparison function for sort
// By Guiming @ 2016-09-04
int compare ( const void *pa, const void *pb )
{
    const float *a = (const float *)pa;
    const float *b = (const float *)pb;
    if(a[0] == b[0])
        return a[1] - b[1];
    else
        return a[0] > b[0] ? 1 : -1;
}

void SortSamplePoints(SamplePoints Points) {
	const int n = 100;
	SamplePoints temPoints = CopySamplePoints(Points);

	float distances[n][2];
	for (int i = 0; i < n; i++)
	{
		distances[i][0] = Points.distances[i];
		distances[i][1] = i * 1.0f;
	}
	/*
	for(int i = 0; i < n; ++i)
	  printf("%.1f\n", Points.distances[i]);
  printf("\n");
	*/

	qsort(distances, n, sizeof(distances[0]), compare);

	for (int i = 0; i < n; i++)
	{
		int idx = (int)distances[i][1];
		Points.xCoordinates[i] = temPoints.xCoordinates[idx];
		Points.yCoordinates[i] = temPoints.yCoordinates[idx];
		Points.weights[i] = temPoints.weights[idx];
		Points.distances[i] = temPoints.distances[idx];
	}
	/*
	for(int i = 0; i < n; ++i)
	  printf("%.1f\n", Points.distances[i]);
	*/
	FreeSamplePoints(&temPoints);

}

// build a KDtree on sample points
// By Guiming @ 2016-09-07
void BuildCPUKDtree (SamplePoints Points){
	int NPTS = Points.numberOfPoints;
	dataP = vector<Point>(NPTS);
	for(int i = 0; i < NPTS; i++){
		dataP[i].coords[0] = Points.xCoordinates[i];
    dataP[i].coords[1] = Points.yCoordinates[i];
	}
	int max_level = (int)(log(dataP.size())/log(2) / 2) + 1;
	tree.Create(dataP, max_level);
}

void BuildGPUKDtree ()
{
	for(int i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);
		GPU_tree[i].CreateKDTree(tree.GetRoot(), tree.GetNumNodes(), dataP);	
	}
	hipSetDevice(0);
}

//Enable P2P Access Across Devices
//Timothy @ 08/13/2020
void EnableP2P()
{
	hipError_t error = hipSuccess;
	for (int id = 0; id < GPU_N; ++id)
	{
		hipSetDevice(id);
		const int top = id > 0 ? id - 1 : (GPU_N - 1); //Int representing first in list of GPUs
		int capable = 1; //(T/F) P2P Access is enabled between devices 
		error = hipDeviceCanAccessPeer(&capable, id, top);
		if (error != hipSuccess)
		{
			printf("ERROR 1 in EnableP2P: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		if (capable)
		{
			printf("Enabled P2P for Device %d...\n", id);
			hipDeviceEnablePeerAccess(top, 0);
		}
		else if (!capable){printf("NOT CAPABLE! P2P for Device %d...\n", id);}
		const int bottom = (id + 1) % GPU_N;
		if (top != bottom)
		{
			error = hipDeviceCanAccessPeer(&capable, id, bottom);
			if (error != hipSuccess)
			{
				printf("ERROR 2 in EnableP2P: %s\n", hipGetErrorString(error));
				exit(EXIT_FAILURE);
			}
			if (capable)
			{
				printf("Enabling P2P for Device %d...\n", id);
				hipDeviceEnablePeerAccess(bottom, 0);
			}
			else if (!capable){printf("NOT CAPABLE! P2P for Device %d...\n", id);}
		}
	}
	hipSetDevice(0); //Reset device to first GPU
}

//By Timothy @ 08/14/2020
//Determine next Device to be used based on passed integers assumed to represent their numbers
void nextDev(int numDev, int& curDev)
{
	if (curDev == (numDev - 1))
	{
		curDev = 0;
	}
	else
	{
		curDev++;
	}
}

//Timothy @ 08/24/2020
//Function to check device properties, primarily for troubleshooting purposes
void DevProp()
{
	for (int i = 0; i < GPU_N; i++)
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
	}
}

////Timothy @ 12/29/20
////Function which copies each group of points into a temorary place on the host, before copying their values to
////hPoints in order to reform the original group
//void ReformPoints(SamplePoints hPoints, SamplePoints* dPoints)
//{
//	int n = hPoints.numberOfPoints; //Number of TOTAL points
//	int rem = n % GPU_N; //Remainder to determine if number of GPUs divides Number of Points evenly
//	int div = n / GPU_N; //Division of points to be divided amongst GPUs
//	int size; //Size of data chunk being copied to tempPoints
//	int index = 0; //Index for the points we are reforming into
//
//	size = div * sizeof(float);
//
//	hipError_t error = hipSuccess;
//
//	SamplePoints tempPoints = AllocateSamplePoints((div + rem));
//	if (error != hipSuccess)
//	{
//		printf("ERROR 0 in ReformPoints: %s\n", hipGetErrorString(error));
//		exit(EXIT_FAILURE);
//	}
//
//	for (int device = 0; device < GPU_N; device++)
//	{
//		hipSetDevice(device);
//		//If on last GPU, check if GPU_N divided into points evenly (rem==0) 
//		//if not add remainder to size on final GPU
//		if ((device == GPU_N - 1) && (rem != 0))
//		{
//			div += rem;
//		}
//
//		int NBLOCK_W = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
//		int GRID_SIZE_W = (int)(sqrtf(NBLOCK_W)) + 1;
//		dim3 dimGrid_W(GRID_SIZE_W, GRID_SIZE_W);
//		hipSetDevice(device);
//		printf("%d:Points...\n", device); //DEBUGGING
//		PrintPoints << <dimGrid_W, BLOCK_SIZE, 0, streams[device] >> > (dPoints[device], 100);
//		hipStreamSynchronize(streams[device]);
//	
//		//Copy all data from chunk to tempPoints
//		error = hipMemcpy(tempPoints.xCoordinates, dPoints[device].xCoordinates, size, hipMemcpyDeviceToHost);
//		if (error != hipSuccess)
//		{
//			printf("ERROR 1 in ReformPoints: %s\n", hipGetErrorString(error));
//			exit(EXIT_FAILURE);
//		}
//		error = hipMemcpy(tempPoints.xCoordinates, dPoints[device].xCoordinates, size, hipMemcpyDeviceToHost);
//		if (error != hipSuccess)
//		{
//			printf("ERROR 2 in ReformPoints: %s\n", hipGetErrorString(error));
//			exit(EXIT_FAILURE);
//		}
//		error = hipMemcpy(tempPoints.xCoordinates, dPoints[device].xCoordinates, size, hipMemcpyDeviceToHost);
//		if (error != hipSuccess)
//		{
//			printf("ERROR 3 in ReformPoints: %s\n", hipGetErrorString(error));
//			exit(EXIT_FAILURE);
//		}
//		// By Guiming @ 2016-09-02
//		error = hipMemcpy(tempPoints.xCoordinates, dPoints[device].xCoordinates, size, hipMemcpyDeviceToHost);
//		if (error != hipSuccess)
//		{
//			printf("ERROR 4 in ReformPoints: %s\n", hipGetErrorString(error));
//			exit(EXIT_FAILURE);
//		}
//
//		//Loop to merge copied chunk of points into hPoints
//		for (int i = 0; i < hPoints.numberOfPoints; i++)
//		{
//			hPoints.xCoordinates[index] = tempPoints.xCoordinates[index];
//			hPoints.yCoordinates[index] = tempPoints.yCoordinates[index];
//			hPoints.weights[index] = tempPoints.weights[index];
//			hPoints.distances[index] = tempPoints.distances[index];
//			index++;
//		}
//	}
//	hipSetDevice(0); //Reset device to first GPU
//	//Free temp points
//	FreeSamplePoints(&tempPoints);
//}

//Timothy @ 08/13/2021
//Reforms points using indeces rather than actually changing any memory
//We realized when reforming other data values that each GPU already has copies of the full set of point structs
void ReformPoints(SamplePoints* dPoints)
{
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		dPoints[i].numberOfPoints = sPoints.numberOfPoints;
		dPoints[i].start = 0;
		dPoints[i].end = 100;
	}
	hipSetDevice(0);
}

//Timothy @ 08/13/2021
//Divides points using indeces rather than actually changing any memory
void DividePoints(SamplePoints* dPoints)
{
	int n = sPoints.numberOfPoints; //Number of TOTAL points
	int rem = n % GPU_N; //Remainder to determine if number of GPUs divides Number of Points evenly
	int div = n / GPU_N; //Division of points to be divided amongst GPUs
	int index = 0; //Index to track start of each data division
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		//If on last GPU, check if GPU_N divided into points evenly (rem==0) 
		//if not add remainder to size on final GPU
		if ((i == GPU_N - 1) && (rem != 0))
		{
			div += rem;
		}
		dPoints[i].numberOfPoints = div;
		dPoints[i].start = index; //Begin tracking division of points
		index += div; //Add division size to index
		dPoints[i].end = index; //Tracking end of division
	}
	hipSetDevice(0);
}

//Timothy @ 08/10/2021
//Reform density arrays on host and copy back accross devices
void ReformDensities(float** dDen, float* hDen)
{
	int n = sPoints.numberOfPoints; //Number of TOTAL points
	int rem = n % GPU_N; //Remainder to determine if number of GPUs divides Number of Points evenly
	int div = n / GPU_N; //Division of points to be divided amongst GPUs
	int size; //Size of data chunk being copied to tempPoints
	int index = 0; //Index for the points we are reforming into
	hipError_t error = hipSuccess;
	float* tempDen = (float*)malloc(n * sizeof(float));
	size = n * sizeof(float);
	
	for (int device = 0; device < GPU_N; device++)
	{
		hipSetDevice(device);
		//If on last GPU, check if GPU_N divided into points evenly (rem==0) 
		//if not add remainder to size on final GPU
		if ((device == GPU_N - 1) && (rem != 0))
		{
			div += rem;
		}
		
		//Copy all data from chunk to tempPoints
		error = hipMemcpy(tempDen, dDen[device], size, hipMemcpyDeviceToHost);
		if (error != hipSuccess)
		{
			printf("ERROR 1.%d in ReformDensities: %s\n", device, hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		//Loop to merge copied chunk of points into hPoints
		for (int i = 0; i < div; i++)
		{
			hDen[index] = tempDen[index];
			index++;
		}
	}
	//Copy reformed dDen accross GPUs
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		error = hipMemcpy(dDen[i], hDen, size, hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			printf("ERROR 2 in ReformDensities: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
	}
	//Cleanup
	hipSetDevice(0); //Reset device to first GPU
	//Free temp points
	FreeDen(tempDen);
}

//Timothy @ 08/13/2021
//Reform bandwidth arrays on host and copy back accross devices
void ReformBandwidths(float** dBand, float* hBand) 
{
	int n = sPoints.numberOfPoints; //Number of TOTAL points
	int rem = n % GPU_N; //Remainder to determine if number of GPUs divides Number of Points evenly
	int div = n / GPU_N; //Division of points to be divided amongst GPUs
	int size; //Size of data chunk being copied to tempPoints
	int index = 0; //Index for the points we are reforming into
	hipError_t error = hipSuccess;
	float* tempBand = (float*)malloc(n * sizeof(float));
	size = n * sizeof(float);

	for (int device = 0; device < GPU_N; device++)
	{
		hipSetDevice(device);
		//If on last GPU, check if GPU_N divided into points evenly (rem==0) 
		//if not add remainder to size on final GPU
		if ((device == GPU_N - 1) && (rem != 0))
		{
			div += rem;
		}

		//Copy all data from chunk to tempPoints
		error = hipMemcpy(tempBand, dBand[device], size, hipMemcpyDeviceToHost);
		if (error != hipSuccess)
		{
			printf("ERROR 1.%d in ReformBandwidths: %s\n", device, hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		//Loop to merge copied chunk of points into hPoints
		for (int i = 0; i < div; i++)
		{
			hBand[index] = tempBand[index];
			index++;
		}
	}
	//Copy reformed dDen accross GPUs
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		error = hipMemcpy(dBand[i], hBand, size, hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			printf("ERROR 2 in ReformBandwidths: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
	}
	//Cleanup
	hipSetDevice(0); //Reset device to first GPU
	//Free temp bands
	FreeBandwidths(tempBand);
}

//Timothy @ 08/13/2021
//Reform EC Weight arrays on host and copy back accross devices
void ReformECWeights(float** dWeights, float* hWeights)
{
	int n = sPoints.numberOfPoints; //Number of TOTAL points
	int rem = n % GPU_N; //Remainder to determine if number of GPUs divides Number of Points evenly
	int div = n / GPU_N; //Division of points to be divided amongst GPUs
	int size; //Size of data chunk being copied to tempPoints
	int index = 0; //Index for the points we are reforming into
	hipError_t error = hipSuccess;
	float* tempWeights = (float*)malloc(n * sizeof(float));
	size = n * sizeof(float);

	for (int device = 0; device < GPU_N; device++)
	{
		hipSetDevice(device);
		//If on last GPU, check if GPU_N divided into points evenly (rem==0) 
		//if not add remainder to size on final GPU
		if ((device == GPU_N - 1) && (rem != 0))
		{
			div += rem;
		}

		//Copy all data from chunk to tempPoints
		error = hipMemcpy(tempWeights, dWeights[device], size, hipMemcpyDeviceToHost);
		if (error != hipSuccess)
		{
			printf("ERROR 1.%d in ReformBandwidths: %s\n", device, hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		//Loop to merge copied chunk of points into hPoints
		for (int i = 0; i < div; i++)
		{
			hWeights[index] = tempWeights[index];
			index++;
		}
	}
	//Copy reformed dDen accross GPUs
	for (int i = 0; i < GPU_N; i++)
	{
		hipSetDevice(i);
		error = hipMemcpy(dWeights[i], hWeights, size, hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			printf("ERROR 2 in ReformBandwidths: %s\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
	}
	//Cleanup
	hipSetDevice(0); //Reset device to first GPU
	//Free temp weights
	FreeEdgeCorrectionWeights(tempWeights);
}